/*

   GPU fault simulator for TRAX fault model
   Matthew Beckler
   18-645 How to Write Fast Code
   Carnegie Mellon University
   Fall 2012

   Last Updated: March 22, 2017
   Copyright (c) 2013-2017, Matthew Beckler

   This program is free software; you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation; either version 2 of the License, or
   (at your option) any later version.

   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License along
   with this program; if not, write to the Free Software Foundation, Inc.,
   51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.


   Usage: ./fault_sim basename
   Looks for the files basename/basename.easy and basename/basename.tests and TODO
   Stores dictionary output in dictionary.cuda

   For more information about this GPU fault simulator:

   M. Beckler and R. D. S. Blanton, "GPU-accelerated fault dictionary generation for the TRAX fault model," 2017 International Test Conference in Asia (ITC-Asia), Taipei, 2017, pp. 34-39. - doi: 10.1109/ITC-ASIA.2017.8097107 - https://doi.org/10.1109/ITC-ASIA.2017.8097107

   M. Beckler and R. D. Blanton, "Fault simulation acceleration for TRAX dictionary construction using GPUs," 2017 IEEE International Test Conference (ITC), Fort Worth, TX, 2017, pp. 1-9. - doi: 10.1109/TEST.2017.8242078 - https://doi.org/10.1109/TEST.2017.8242078

   For more information about the TRAX fault model:

   M. Beckler and R. D. Blanton, "On-Chip Diagnosis of Generalized Delay Failures using Compact Fault Dictionaries," in IEEE Transactions on Computer-Aided Design of Integrated Circuits and Systems, vol. PP, no. 99, pp. 1-1. - doi: 10.1109/TCAD.2018.2803621 - https://doi.org/10.1109/TCAD.2018.2803621

   Beckler, M. and Blanton, R.D., "On-Chip Diagnosis for Early-Life and Wear-Out Failures," IEEE International Test Conference, Nov. 2012. - https://doi.org/10.1109/TEST.2012.6401580


   TODO list of things to improve:

   * Figure out what's going on with dev_gates, didn't we put this in constants memory?
        Nope, only the gate eval LuT is in constants memory. There's only 64k of constants mem, and we need 128 bytes for the gate eval LuT. Each Gate takes up 12 bytes (and we could only reduce that by limiting us to 2^16 = 65536 nets which might not be ok) so we can have a maximum of 5450 gates, which is not enough. Unless we could use multiple kernels for each chunk of 5450 gates? Probably not worth it.

   * Maybe use shared memory for the netlist information?
        Probably not enough room for the larger circuits we want to handle

   * Store dictionary in memory/disk as binary format?

   * Some way to do multiple gate evals per thread? Maybe a packed data structure? Maybe machine word-width algebraic operations like in that one DAC submission?

   * As mentioned above, some data storage techniques had to be migrated to a compacted storage technique, where multiple 1 or 2 bit values are stored in a single byte. This reduces the total storage required, but makes each access more complex, usually involving one or more bit shift and bitwise masking operations. It would be good to analyze the actual access patterns for these memories, and determine if functions such as "extract four consecutive values" or "extract both v1 and v2 for a given net" would be useful and faster than individual accesses. Additionally, for smaller circuits or on systems with more available memory, it would be useful to be able to automatically detect the memory size and switch between data storage formats based on the system details.

   * There are a number of small optimizations we would like to investigate with regards to array-of-struct vs struct-of-arrays, especially in our gate and test memory storage.

   Update log:
   * Ignore the skip-ahead optimization for Kernel 3, and the N invocations of Kernel 3, by doing one giant Kernel 3 invocation.
   * Support tracking activity of each net (and therefore PMOS input) over the entire test set - Mar 2017
   * Added support for disabling TRAX hazard activation - Mar 2017
   * Added support for TF faults in addition to just TRAX faults - Aug 2013

*/

#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <math.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

/* Here is the "easy" to parse file format for the circuit netlist:
    NUMINPUTS 5
    INPUT 8
    INPUT 9
    INPUT 10
    INPUT 11
    INPUT 12
    NUMOUTPUTS 4
    OUTPUT 6
    OUTPUT 7
    OUTPUT 1
    OUTPUT 0
    NUMGATES 8
    1 0 9 9
    0 1 8 8
    2 2 8 10
    3 3 10 11
    4 4 9 3
    5 5 3 12
    8 6 2 4
    9 7 4 5

    Nets are integers.
    Gates can only have two inputs (buf/inv should have the same net listed as both inputs, the second will be ignored, but won't break the parser).
    GATES ABSOLUTELY MUST BE TOPOLOGICALLY SORTED based on their level within the circuit.
    Gates' output net ids must start at 0.
    There are probably other implicit assumptions that I have forgotten about. Caveat emptor
*/

#define uchar unsigned char
#define uint unsigned int

// This is the definition of all eight supported gate types. Ordering is to make the GPU code more efficient (and/nand/or/nor are commonly grouped together, same with xor/xnor)
#define TYPE_AND    (0)
#define TYPE_NAND   (1)
#define TYPE_OR     (2)
#define TYPE_NOR    (3)
#define TYPE_BUF    (4)
#define TYPE_INV    (5)
#define TYPE_XOR    (6)
#define TYPE_XNOR   (7)

// Since we have a four-valued logic, we need two bits to represent each one.
// I was originally trying to be creative about these assignments, to make it so that we could do gate evaluations in parallel using bit-operation instructions.
// Which is how I decided to make 0 = 00, 1 = 11, and X and H be the other two.
// However, since we haven't implemented parallel gate evaluations, this is kind of a moot point. However these values are indeed being used.
#define LOGIC_0 (0)
#define LOGIC_X (1)
#define LOGIC_H (2)
#define LOGIC_1 (3)

// How many threads per block? Maximum is 512, make sure each is a multiple of 16, multiple of 32 is probably better.
// Doesn't really seem to affect performance much, so something else is probably limiting the speed.
#define FAULTS_PER_BLOCK_KERNEL_1 (512)
#define FAULTS_PER_BLOCK_KERNEL_2 (512)
#define FAULTS_PER_BLOCK_KERNEL_3 (512)

// FYI, pointers cost 8 bytes!

// Structure to represent a gate
// We should pack in1 and in2 together in the same int if we need to save memory
typedef struct
{
    uchar type;
    uchar is_output;
    uint in1;
    uint in2;
} Gate; // 12 bytes due to nice packing

// Structure to represent a fault
typedef struct
{
    uint net;
    uint polarity; // 0 = falling, 1 = rising
} Fault;


// Used to evaluate how long (wall-clock) the three kernels take to run. Taken from:
// http://stackoverflow.com/questions/1468596/c-programming-calculate-elapsed-time-in-milliseconds-unix
int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1)
{
    long int diff = (t2->tv_usec + 1000000 * t2->tv_sec) - (t1->tv_usec + 1000000 * t1->tv_sec);
    result->tv_sec = diff / 1000000;
    result->tv_usec = diff % 1000000;

    return (diff<0);
}
// Accumulates (end - start) into result
void timeval_accumulate_diff(struct timeval *result, struct timeval *start, struct timeval *end)
{
    long int diff_usec = (end->tv_usec + 1000000 * end->tv_sec) - (start->tv_usec + 1000000 * start->tv_sec);
    diff_usec += result->tv_usec;
    result->tv_sec += diff_usec / 1000000;
    result->tv_usec = diff_usec % 1000000;
}

// Prints a nice number of bytes (KB, MB, GB, etc) to the provided buffer.
void pretty_bytes(char* buf, uint bytes)
{
    const char* suffixes[] = {"B", "KB", "MB", "GB", "TB", "PB", "EB"};
    uint order = 0;
    double count = bytes;
    while ((count >= 1024) && (order < sizeof(suffixes))) {
        order++;
        count /= 1024;
    }
    if ((count - floor(count)) == 0.0) {
        sprintf(buf, "%d %s", (int)count, suffixes[order]);
    } else {
        sprintf(buf, "%.1f %s", count, suffixes[order]);
    }
}


char printable_logic_value(char value)
{
    switch (value)
    {
        case LOGIC_0: return '0';
        case LOGIC_X: return 'X';
        case LOGIC_H: return 'H';
        case LOGIC_1: return '1';
    }
    return '?';
}


// Circuit states are stored as pairs of values (v1, v2) for each net in the circuit.
void print_state(uchar* state, uint num_state_values)
{
    for (uint i = 0; i < num_state_values; i++)
    {
        uchar s = state[i];
        printf("%c", printable_logic_value(s));
        if (i % 2 == 1)
            printf(" ");
    }
    printf("\n");
}
void print_state_raw(uchar* state, uint num_state_values)
{
    for (uint i = 0; i < num_state_values; i++)
    {
        printf("%X", state[i]);
    }
    printf("\n");
}

// For some data like the tests values we pack 8 1-bit values into each byte.
// These are the accessor functions to make it easy to access that data.
__host__ __device__ void BIT_SET_UCHAR(uchar* array, uint which, uchar value)
{
    uchar* b = array + (which / 8);
    uchar shift = which % 8;
    if (value)
        *b |= (1 << shift);
    else
        *b &= ~(1 << shift);
}
__host__ __device__ uchar BIT_GET_UCHAR(uchar* array, uint which)
{
    uchar* b = array + (which / 8);
    uchar shift = which % 8;
    if (*b & (1 << shift))
        return 1;
    else
        return 0;
}

// This is the new gate evaluation lookup table.
// Simply concatenate the gate type with the four bits of input values.
__constant__ uchar gate_eval_lut[128] = {
    // gate 000 (and)
    LOGIC_0, // 0 0 -> 0
    LOGIC_0, // 0 X -> 0
    LOGIC_0, // 0 H -> 0
    LOGIC_0, // 0 1 -> 0
    LOGIC_0, // X 0 -> 0
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X
    LOGIC_X, // X 1 -> X
    LOGIC_0, // H 0 -> 0
    LOGIC_X, // H X -> X
    LOGIC_H, // H H -> H
    LOGIC_H, // H 1 -> H
    LOGIC_0, // 1 0 -> 0
    LOGIC_X, // 1 X -> X
    LOGIC_H, // 1 H -> H
    LOGIC_1, // 1 1 -> 1

    // gate 001 (nand)
    LOGIC_1, // 0 0 -> 1
    LOGIC_1, // 0 X -> 1
    LOGIC_1, // 0 H -> 1
    LOGIC_1, // 0 1 -> 1
    LOGIC_1, // X 0 -> 1
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X
    LOGIC_X, // X 1 -> X
    LOGIC_1, // H 0 -> 1
    LOGIC_X, // H X -> X
    LOGIC_H, // H H -> H
    LOGIC_H, // H 1 -> H
    LOGIC_1, // 1 0 -> 1
    LOGIC_X, // 1 X -> X
    LOGIC_H, // 1 H -> H
    LOGIC_0, // 1 1 -> 0

    // gate 010 (or)
    LOGIC_0, // 0 0 -> 0
    LOGIC_X, // 0 X -> X
    LOGIC_H, // 0 H -> H
    LOGIC_1, // 0 1 -> 1
    LOGIC_X, // X 0 -> X
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X
    LOGIC_1, // X 1 -> 1
    LOGIC_H, // H 0 -> H
    LOGIC_X, // H X -> X
    LOGIC_H, // H H -> H
    LOGIC_1, // H 1 -> 1
    LOGIC_1, // 1 0 -> 1
    LOGIC_1, // 1 X -> 1
    LOGIC_1, // 1 H -> 1
    LOGIC_1, // 1 1 -> 1

    // gate 011 (nor)
    LOGIC_1, // 0 0 -> 1
    LOGIC_X, // 0 X -> X
    LOGIC_H, // 0 H -> H
    LOGIC_0, // 0 1 -> 0
    LOGIC_X, // X 0 -> X
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X
    LOGIC_0, // X 1 -> 0
    LOGIC_H, // H 0 -> H
    LOGIC_X, // H X -> X
    LOGIC_H, // H H -> H
    LOGIC_0, // H 1 -> 0
    LOGIC_0, // 1 0 -> 0
    LOGIC_0, // 1 X -> 0
    LOGIC_0, // 1 H -> 0
    LOGIC_0, // 1 1 -> 0

    // gate 100 (buffer)
    LOGIC_0, // 0 0 -> 0
    LOGIC_0, // 0 X -> 0 - not possible, for BUF/INV must have same values
    LOGIC_0, // 0 H -> 0 - not possible, for BUF/INV must have same values
    LOGIC_0, // 0 1 -> 0 - not possible, for BUF/INV must have same values
    LOGIC_X, // X 0 -> X - not possible, for BUF/INV must have same values
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X - not possible, for BUF/INV must have same values
    LOGIC_X, // X 1 -> X - not possible, for BUF/INV must have same values
    LOGIC_H, // H 0 -> H - not possible, for BUF/INV must have same values
    LOGIC_H, // H X -> H - not possible, for BUF/INV must have same values
    LOGIC_H, // H H -> H
    LOGIC_H, // H 1 -> H - not possible, for BUF/INV must have same values
    LOGIC_1, // 1 0 -> H - not possible, for BUF/INV must have same values
    LOGIC_1, // 1 X -> H - not possible, for BUF/INV must have same values
    LOGIC_1, // 1 H -> H - not possible, for BUF/INV must have same values
    LOGIC_1, // 1 1 -> H

    // gate 001 (inverter)
    LOGIC_1, // 0 0 -> 1
    LOGIC_1, // 0 X -> 1 - not possible, for BUF/INV must have same values
    LOGIC_1, // 0 H -> 1 - not possible, for BUF/INV must have same values
    LOGIC_1, // 0 1 -> 1 - not possible, for BUF/INV must have same values
    LOGIC_X, // X 0 -> X - not possible, for BUF/INV must have same values
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X - not possible, for BUF/INV must have same values
    LOGIC_X, // X 1 -> X - not possible, for BUF/INV must have same values
    LOGIC_H, // H 0 -> H - not possible, for BUF/INV must have same values
    LOGIC_H, // H X -> H - not possible, for BUF/INV must have same values
    LOGIC_H, // H H -> H
    LOGIC_H, // H 1 -> H - not possible, for BUF/INV must have same values
    LOGIC_0, // 1 0 -> 0 - not possible, for BUF/INV must have same values
    LOGIC_0, // 1 X -> 0 - not possible, for BUF/INV must have same values
    LOGIC_0, // 1 H -> 0 - not possible, for BUF/INV must have same values
    LOGIC_0, // 1 1 -> 0

    // gate 110 (xor)
    LOGIC_0, // 0 0 -> 0
    LOGIC_X, // 0 X -> X
    LOGIC_H, // 0 H -> H
    LOGIC_1, // 0 1 -> 1
    LOGIC_X, // X 0 -> X
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X
    LOGIC_X, // X 1 -> X
    LOGIC_H, // H 0 -> H
    LOGIC_X, // H X -> X
    LOGIC_H, // H H -> H
    LOGIC_H, // H 1 -> H
    LOGIC_1, // 1 0 -> 1
    LOGIC_X, // 1 X -> X
    LOGIC_H, // 1 H -> H
    LOGIC_0, // 1 1 -> 0

    // gate 111 (xnor)
    LOGIC_1, // 0 0 -> 1
    LOGIC_X, // 0 X -> X
    LOGIC_H, // 0 H -> H
    LOGIC_0, // 0 1 -> 0
    LOGIC_X, // X 0 -> X
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X
    LOGIC_X, // X 1 -> X
    LOGIC_H, // H 0 -> H
    LOGIC_X, // H X -> X
    LOGIC_H, // H H -> H
    LOGIC_H, // H 1 -> H
    LOGIC_0, // 1 0 -> 0
    LOGIC_X, // 1 X -> X
    LOGIC_H, // 1 H -> H
    LOGIC_1, // 1 1 -> 1
};


// This is the core cuda fault simulation function, only called from the two fault sim wrapper functions (kernels 1 and 3).
// It will probably be inlined into each kernel by the compiler, but this is better for future maintenance
// Inputs:
//      gates           The global, unchanging gate structure
//      my_state        Pointer to the state for this simulation (explained above, basically it's just net0_v1, net0_v2, net1_v1, net1_v2, etc)
// Outputs:
//      Updates my_state in place with the correct values after fault simulation
// Returns:
//      The v2 value
#define TRAN0110 ( (LOGIC_0 << 6) | (LOGIC_1 << 4) | (LOGIC_1 << 2) | LOGIC_0 )
#define TRAN1001 ( (LOGIC_1 << 6) | (LOGIC_0 << 4) | (LOGIC_0 << 2) | LOGIC_1 )
#define TRAN01H0 ( (LOGIC_0 << 6) | (LOGIC_1 << 4) | (LOGIC_H << 2) | LOGIC_0 )
#define TRAN100H ( (LOGIC_1 << 6) | (LOGIC_0 << 4) | (LOGIC_0 << 2) | LOGIC_H )
#define TRAN011H ( (LOGIC_0 << 6) | (LOGIC_1 << 4) | (LOGIC_1 << 2) | LOGIC_H )
#define TRAN10H1 ( (LOGIC_1 << 6) | (LOGIC_0 << 4) | (LOGIC_H << 2) | LOGIC_1 )
__device__ uchar cuda_fault_sim_core(const Gate* g, uchar* my_state, uint gate_id)
{
    // could we transpose our state matrix to make the accesses align better? TODO
    // Each thread is accessing the same net at the same time, maybe we could make those accesses be coallesced?
    uchar in1_v1 = 0x7F & my_state[g->in1 * 2];
    uchar in2_v1 = 0x7F & my_state[g->in2 * 2];
    uchar in1_v2 = 0x7F & my_state[g->in1 * 2 + 1];
    uchar in2_v2 = 0x7F & my_state[g->in2 * 2 + 1];

    // Using the new gate evaluation lookup table, we use the gate type and two input values to craft a 7-bit index
    uchar v1 = gate_eval_lut[(g->type << 4) | (in1_v1 << 2) | (in2_v1)];
    uchar v2 = gate_eval_lut[(g->type << 4) | (in1_v2 << 2) | (in2_v2)];

#if (USE_TRAX && USE_HAZARDS)
    // Now we need to detect if v2 should result in a hazard due to this gate's inputs
    uchar merged_values = (in1_v1 << 6) | (in2_v1 << 4) | (in1_v2 << 2) | in2_v2;
    // and/nand = 0/1, or/nor = 2/3
    uchar hazard_and_nand = (g->type <= 1) &&                 (merged_values == TRAN0110 || merged_values == TRAN1001 || merged_values == TRAN01H0 || merged_values == TRAN100H);
    uchar hazard_or_nor   = (g->type <= 3 && g->type >= 2) && (merged_values == TRAN0110 || merged_values == TRAN1001 || merged_values == TRAN011H || merged_values == TRAN10H1);
    uchar hazard_xor_xnor = (g->type >  5) && (v1 == v2 && in1_v1 != in1_v2 && in2_v1 != in2_v2);
    uchar hazard = hazard_and_nand || hazard_or_nor || hazard_xor_xnor;
    v2 = hazard ? LOGIC_H : v2;
#endif

    // Update my_state in place with the (potentially new) values of v1 and v2:
    // Note that unlike the reference implementation, we don't care if the values were updated, since we check all downstream gates regardless.
    my_state[gate_id * 2] = v1;
    my_state[gate_id * 2 + 1] = v2;

    return v2;
}


// This is Kernel 1, the cuda fault-free fault simulation function. Pass in the gates, all the states (one per test), the state size, and the number of gates.
// Updates all the states in place. One thread per test pair, FAULTS_PER_BLOCK_KERNEL_1 threads per block.
__global__ void cuda_fault_free_fault_sim(Gate* gates, uint num_gates, uchar* all_states, uint state_bytes, uint num_tests)
{
    // each thread will go through a complete fault simulation
    // since we are not skipping around, all threads stay in lock-step
    // Key idea: branching is ok as long as all the threads do the same thing
    // This means we can't have any data-dependent branching
    // Since each thread evaluates the same gate at the same time, that kind of branching is ok
    // However we can't have branching based on the actual logic values going through the gate

    uint test_id = (blockIdx.x * FAULTS_PER_BLOCK_KERNEL_1) + threadIdx.x;
    if (test_id < num_tests)
    {
        uchar* my_state = all_states + (state_bytes * test_id);
        // We iterate over all the relevant gates, in topological order (this sorting was already handled, our netlist comes pre-sorted)
        for (uint gate_id = 0; gate_id < num_gates; gate_id++)
        {
            cuda_fault_sim_core(&gates[gate_id], my_state, gate_id);
        }
    }
}

// This is Kernel 2, which determines which tests activate each fault. One thread per fault.
// It stores a 0 or 1 into the matrix fault_activation, which now uses a packed format (one bit for every fault-test pair), each row is a fault, each col is a test. (initialized to all zeros).
__global__ void cuda_check_fault_activations(Gate* gates, uchar* all_states, uint state_bytes, uint num_tests, uchar* fault_activations, uint num_faults)
{
    uint fault_id = blockIdx.x * FAULTS_PER_BLOCK_KERNEL_2 + threadIdx.x;
    if (fault_id < num_faults)
    {
        uint gate_id = fault_id / 2;
        uint rising = (fault_id % 2); // rising when fault_id is odd
        // this thread has to check for fault activation for fault_id for all tests
        for (uint test_id = 0; test_id < num_tests; test_id++)
        {
            uchar* my_state = all_states + test_id * state_bytes;
            uchar v1 = my_state[gate_id * 2];
            uchar v2 = my_state[gate_id * 2 + 1];
            uchar activated = ( ( rising && (v1 == LOGIC_0) && (v2 == LOGIC_1)) || // output transition activation
                                (!rising && (v1 == LOGIC_1) && (v2 == LOGIC_0)) || // ditto
                                (v2 == LOGIC_H) );                                 // hazard-based activation - And actually, if we are not doing TRAX fault sim, then there can be no LOGIC_H values, so this activation condition is harmless, neat!
            // argh, this is also susceptible to the problem of multiple threads writing to the same byte concurrently
            //BIT_SET_UCHAR(fault_activations, fault_id * num_tests + test_id, (activated ? 1 : 0));
            // (TODO - This comment is talking about BIT_SET_UCHAR, right? Not the atomicOr, right?)

            uint index = fault_id * num_tests + test_id;
            atomicOr( ((uint*)fault_activations) + (index / 32), activated << (index % 32));
        }
    }
}


// This is Kernel 3, which does the faulty fault simulation. One grid of blocks per fault, FAULTS_PER_BLOCK_KERNEL_3 threads per block, one thread per activating test.
// NEW PLAN: Update all the states in place, but never copy the faulty states back to the CPU.
// Instead, we now have the dictionary row in memory and we directly write the pass/fail bit into that memory directly, and then copy it back to the CPU and write it to disk.
__global__ void cuda_faulty_fault_sim(Gate* dev_gates, uint num_gates, uchar* dev_faulty_states, uint state_bytes, uint total_activations)
{
    uint offset = blockIdx.x * FAULTS_PER_BLOCK_KERNEL_3 + threadIdx.x;
    if (offset < total_activations)
    {
        uchar* my_state = dev_faulty_states + state_bytes * offset;
        // Removed fault activation since it now happens on the CPU
        uchar test_failed = 0; // Local copy since we'll be writing it many times, copy it to final byte of my_state at the end

        // We iterate over all the relevant gates, in topological order (this sorting was already handled, our netlist comes pre-sorted)
        for (uint gate_id = 0; gate_id < num_gates; gate_id++)
        {
            Gate *g = &dev_gates[gate_id]; // should be the same for all threads in the kernel
            uchar v2 = my_state[gate_id * 2 + 1]; // Existing V2
            if (!(v2 & 0x80)) {
                // 0x80 set = "activated fault site" so we don't re-simulate it and un-activate the fault!
                v2 = cuda_fault_sim_core(g, my_state, gate_id);
            }

        #if USE_TRAX
            test_failed = (g->is_output && ((0x7F & v2) == LOGIC_X)) ? 1 : test_failed; // If an X reaches an output, the test fails
        #else
            test_failed = (g->is_output && (v2 != my_state[gate_id * 2 + 1])) ? 1 : test_failed; // If an output does not match the expected (fault-free) value, the test fails
        #endif
        }
        my_state[state_bytes - 1] = test_failed;
    }
}

void check_cuda_errors(const char *kernel_name)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA error in kernel %s: %s\n", kernel_name, hipGetErrorString(err));
        exit(1);
    }
}

// does an integer division of n/d, rounding up
uint divide_round_up(uint n, uint d)
{
    return (n + (d - 1)) / d;
}

// TODO use compile-time directives for the TRAX vs TF code to improve performance?

// This is our main function.
// It would be stellar to split the parsing code into a separate function in a separate file.
int main(int argc, char* argv[])
{
    char buffer[100];  // for using pretty_bytes(buffer, numbytes);

    // Track how many bytes we malloc on the CPU and the CUDA device
    unsigned long malloc_bytes = 0;
    unsigned long cuda_malloc_bytes = 0;

    // I added some timing instrumentation in the code for different sections
    struct timeval tvStart, tvDoneParsing, tvPreK1, tvPostK1, tvPreK2, tvPostK2, tvPreK3, tvPostK3, tvEnd, tvDiff;
    gettimeofday(&tvStart, NULL);

    printf("This code built with these recognized options:\n"
#if USE_TRAX
           "USE_TRAX "
#   if USE_HAZARDS
           "USE_HAZARDS "
#   else
           "NO_HAZARDS "
#   endif
#else
           "USE_TF "
#endif
           "\n");

    if (argc != 2)
    {
        fprintf(stderr, "Usage: %s basename}\n", argv[0]);
        exit(1);
    }

    char* basename = argv[1]; // something like "c432"

    malloc_bytes += (2 * strlen(basename) + 6 + 1);
    char* filename_input = (char*) malloc(2 * strlen(basename) + 6 + 1); // "c432/c432.v", so 2*N + 6 for "/.easy" + 1 for the '\0'
    assert(filename_input != NULL);
    sprintf(filename_input, "%s/%s.easy", basename, basename);
    printf("Netlist filename: '%s'\n", filename_input);

    malloc_bytes += (2 * strlen(basename) + 12 + 1);
    char* filename_tests = (char*) malloc(2 * strlen(basename) + 12 + 1); // "c432/c432.tests.easy", so 2*N + 12 for "/.tests.easy" + 1 for the '\0'
    assert(filename_tests != NULL);
    sprintf(filename_tests, "%s/%s.tests.easy", basename, basename);
    printf("Tests filename: '%s'\n", filename_tests);

    malloc_bytes += (2 * strlen(basename) + 27 + 1);
    char* filename_dictionary = (char*) malloc(2 * strlen(basename) + 27 + 1); // "c432/c432.dictionary.traxnh.pf.cuda", so 2*N + 27 for "/.dictionary.traxnh.pf.cuda" + 1 for the '\0'
    assert(filename_dictionary != NULL);
    sprintf(filename_dictionary,
#if USE_TRAX
#   if USE_HAZARDS
    "%s/%s.dictionary.trax.pf.cuda",
#   else
    "%s/%s.dictionary.traxnh.pf.cuda",
#   endif
#else
    "%s/%s.dictionary.tf.pf.cuda",
#endif
    basename, basename);
    printf("Dictionary filename: '%s'\n", filename_dictionary);

    malloc_bytes += (2 * strlen(basename) + 13 + 1);
    char* filename_faults = (char*) malloc(2 * strlen(basename) + 13 + 1); // "c432/c432.faults.gpu", so 2*N + 13 for "/.faults.gpu" + 1 for the '\0'
    assert(filename_faults != NULL);
    sprintf(filename_faults, "%s/%s.faults.gpu", basename, basename);
    printf("Faults filename: '%s'\n", filename_faults);

    malloc_bytes += (2 * strlen(basename) + 7 + 1);
    char* filename_usage = (char*) malloc(2 * strlen(basename) + 7 + 1); // "c432/c432.usage", so 2*N + 7 for "/.usage" + 1 for the '\0'
    assert(filename_usage != NULL);
    sprintf(filename_usage, "%s/%s.usage", basename, basename);
    printf("Usage filename: '%s'\n", filename_usage);

    // Circuit netlist data
    uint num_inputs, num_outputs, num_gates;
    uint* inputs;
    uint* outputs;
    Gate* gates;

    // Test patterns data
    uint num_tests;
    uchar *tests_v1;
    uchar *tests_v2;
    uchar *tests_expected;

    // Fault data
    uint num_faults;
    Fault* faults;

    // Ideally, we could just now say something like:
    // read_netlist(&num_inputs, &num_outputs, &num_gates, &num_faults, &inputs, &outputs, &gates);
    // read_tests(&num_tests, &tests);
    // ------------- Begin "please move to separate function/file" section --------------

    // now we need to load in our circuit netlist
    FILE* fp = fopen(filename_input, "r");
    assert(fp != NULL);

    if (fscanf(fp, "NUMINPUTS %d\n", &num_inputs) != 1)
    {
        fprintf(stderr, "Unable to parse NUMINPUTS line!\n");
        exit(1);
    }

    printf("Detected %d inputs\n", num_inputs);
    malloc_bytes += (sizeof(uint) * num_inputs);
    inputs = (uint*) malloc(sizeof(uint) * num_inputs);
    assert(inputs != NULL);
    for (uint i = 0; i < num_inputs; i++)
    {
        if(fscanf(fp, "INPUT %d\n", &inputs[i]) != 1)
        {
            fprintf(stderr, "Unable to parse INPUT line #%d!\n", i);
            exit(1);
        }
    }


    if (fscanf(fp, "NUMOUTPUTS %d\n", &num_outputs) != 1)
    {
        fprintf(stderr, "Unable to parse NUMOUTPUTS line!\n");
        exit(1);
    }

    printf("Detected %d outputs\n", num_outputs);
    malloc_bytes += (sizeof(uint) * num_outputs);
    outputs = (uint*) malloc(sizeof(uint) * num_outputs);
    assert(outputs != NULL);
    for (uint i = 0; i < num_outputs; i++)
    {
        if (fscanf(fp, "OUTPUT %d\n", &outputs[i]) != 1)
        {
            fprintf(stderr, "Unable to parse OUTPUT line #%d!\n", i);
            exit(1);
        }
    }


    if (fscanf(fp, "NUMGATES %d\n", &num_gates) != 1)
    {
        fprintf(stderr, "Unable to parse NUMGATES line!\n");
        exit(1);
    }

    // IMPORTANT - We assume that the gates in the file are already in topological order!
    pretty_bytes(buffer, sizeof(Gate) * num_gates);
    printf("Detected %d gates (%s)\n", num_gates, buffer);
    malloc_bytes += (sizeof(Gate) * num_gates);
    gates = (Gate*) malloc(sizeof(Gate) * num_gates);
    uint type, out, in1, in2;
    for (uint i = 0; i < num_gates; i++)
    {

        if (fscanf(fp, "%d %d %d %d\n", &type, &out, &in1, &in2) != 4)
        {
            fprintf(stderr, "Unable to parse gate line #%d!\n", i);
            exit(1);
        }
        // there has to be a better way to do "if i in list of outputs"
        uchar is_output = 0;
        for (uint output_id = 0; output_id < num_outputs; output_id++)
        {
            if (i == outputs[output_id])
            {
                is_output = 1;
                break;
            }
        }
        gates[i].type = type;
        gates[i].is_output = is_output;
        gates[i].in1 = in1;
        gates[i].in2 = in2;
    }
    fclose(fp);


    // Read in tests
    fp = fopen(filename_tests, "r");
    assert(fp != NULL);

    if (fscanf(fp, "NUMTESTS %d\n", &num_tests) != 1)
    {
        fprintf(stderr, "Unable to parse NUMTESTS line!\n");
        exit(1);
    }

    uint size_v1_v2 = divide_round_up(num_inputs, 8);
    uint size_expected = divide_round_up(num_outputs, 8);
    pretty_bytes(buffer, (size_v1_v2 * 2 + size_expected) * num_tests);
    printf("Detected %d tests (%s)\n", num_tests, buffer);

    // NEW PLAN - Information stored in a compacted format, eight bits per byte, no TestPair structure, just big arrays for v1, v2, and expected, since pointers cost us 8 bytes!
    malloc_bytes += (size_v1_v2 * num_tests);
    malloc_bytes += (size_v1_v2 * num_tests);
    malloc_bytes += (size_expected * num_tests);
    tests_v1 = (uchar*) malloc(size_v1_v2 * num_tests);
    tests_v2 = (uchar*) malloc(size_v1_v2 * num_tests);
    tests_expected = (uchar*) malloc(size_expected * num_tests);
    assert(tests_v1 != NULL);
    assert(tests_v2 != NULL);
    assert(tests_expected != NULL);

    // these buffers are just for reading from the file
    malloc_bytes += (num_inputs + 1);
    char* buf_v1 = (char*) malloc(num_inputs + 1);
    assert(buf_v1 != NULL);
    malloc_bytes += (num_inputs + 1);
    char* buf_v2 = (char*) malloc(num_inputs + 1);
    assert(buf_v2 != NULL);
    malloc_bytes += (num_outputs + 1);
    char* buf_expected = (char*) malloc(num_outputs + 1);
    assert(buf_expected != NULL);
    for (uint test_id = 0; test_id < num_tests; test_id++)
    {
        if (fscanf(fp, "%s %s %s\n", buf_v1, buf_v2, buf_expected) != 3)
        {
            fprintf(stderr, "Unable to parse tests line #%d!\n", test_id);
            exit(1);
        }

        // now we need to convert the values to our new special compacted binary format
        for (uint i = 0; i < num_inputs; i++)
        {
            BIT_SET_UCHAR(tests_v1 + size_v1_v2 * test_id, i, (buf_v1[i] == '0') ? 0 : 1);
            BIT_SET_UCHAR(tests_v2 + size_v1_v2 * test_id, i, (buf_v2[i] == '0') ? 0 : 1);
        }
        for (uint i = 0; i < num_outputs; i++)
        {
            BIT_SET_UCHAR(tests_expected, size_expected * test_id + i, (buf_expected[i] == '0') ? 0 : 1);
        }
    }
    free(buf_v1);
    free(buf_v2);
    free(buf_expected);
    fclose(fp);


    // Read in list of faults
    fp = fopen(filename_faults, "r");
    assert(fp != NULL);

    if (fscanf(fp, "NUM_FAULTS %d\n", &num_faults) != 1)
    {
        fprintf(stderr, "Unable to parse NUM_FAULTS line!\n");
        exit(1);
    }

    printf("Detected %d faults\n", num_faults);
    malloc_bytes += (sizeof(Fault) * num_faults);
    faults = (Fault*) malloc(sizeof(Fault) * num_faults);
    assert(faults != NULL);
    uint net, polarity;
    for (uint ix = 0; ix < num_faults; ix++)
    {
        if (fscanf(fp, "%d %d\n", &net, &polarity) != 2)
        {
            fprintf(stderr, "Unable to parse fault line #%d!\n", ix);
            exit(1);
        }

        faults[ix].net = net;
        faults[ix].polarity = polarity;
    }
    fclose(fp);


    printf("Finished parsing files!\n");
    printf("--------------------------------------------\n");

    // ------------- End "please move to separate function/file" section --------------

    gettimeofday(&tvDoneParsing, NULL);

    /***************************************************************************
     * The Grand Plan
     * A single-fault-multiple-pattern approach:
     * 1. First, we run a parallel fault simulation to determine the fault-free
     * circuit state for every test pair.
     * 2. Then, for each fault, we determine which test pairs activate the fault
     * 3. We can ignore test pairs that do not activate the fault.
     * 4. We do another parallel fault simulation on only the patterns that
     *    activate the fault. We also have to change each state to put the X
     *    value at the necessary net before fault simulation.
    ***************************************************************************/

    /*_  ________ _____  _   _ ______ _        __
    | |/ /  ____|  __ \| \ | |  ____| |      /_ |
    | ' /| |__  | |__) |  \| | |__  | |       | |
    |  < |  __| |  _  /| . ` |  __| | |       | |
    | . \| |____| | \ \| |\  | |____| |____   | |
    |_|\_\______|_|  \_\_| \_|______|______|  |_|
    */

    // We need to store the circuit state (v1 and v2 values for all nets) for all tests.
    uint num_nets = num_inputs + num_gates;
    uint num_state_values = num_nets * 2; // times 2, since we have v1 and v2 state! see explanation above
    uint state_bytes = num_state_values + 1; // weird, but it works - Plus 1 for the extra byte at the end to track detected/not-detected.
    uint all_states_size = state_bytes * num_tests; // number of bytes required to store a state for each test
    pretty_bytes(buffer, all_states_size);
    printf("Detected %d nets, requiring %u B per state, %s total\n", num_nets, state_bytes, buffer);

    // we allocate a circuit state for each test, and then we find the fault-free values in the circuit for each test
    malloc_bytes += (all_states_size);
    uchar* fault_free_states = (uchar*) malloc(all_states_size);
    assert(fault_free_states != NULL);
    // let's set the values all to X (X=01, so 01010101, so 0x55) - This uses our new compact format for storing the state information
    memset(fault_free_states, 0x55, all_states_size);
    //print_state(fault_free_states, num_state_values);
    //print_state_raw(fault_free_states, num_state_values);

    // Set the input patterns in all fault-free states
    for (uint test_id = 0; test_id < num_tests; test_id++)
    {
        uchar* this_state = fault_free_states + test_id * state_bytes;

        // Clear the detected byte for this test
        this_state[state_bytes - 1] = 0;

        // set the input values in the fault free states
        for (uint input_id = 0; input_id < num_inputs; input_id++)
        {
            this_state[inputs[input_id] * 2]     = (BIT_GET_UCHAR(tests_v1 + size_v1_v2 * test_id, input_id) == 0 ? LOGIC_0 : LOGIC_1);
            this_state[inputs[input_id] * 2 + 1] = (BIT_GET_UCHAR(tests_v2 + size_v1_v2 * test_id, input_id) == 0 ? LOGIC_0 : LOGIC_1);
        }
    }

    // Now that we have set the input patterns in all the fault-free states, we run the simulations in parallel to find the fault-free circuit states for all tests
    // We run a parallel fault simulation to determine the fault-free circuit state for every test pair

    // the gpu needs copies of the states and the gates list-of-structs
    uchar* dev_fault_free_states;
    cuda_malloc_bytes += all_states_size;
    hipMalloc( (void**)&dev_fault_free_states, all_states_size);
    check_cuda_errors("(hipMalloc dev_fault_free_states)");
    assert(dev_fault_free_states != NULL);
    hipMemcpy( dev_fault_free_states, fault_free_states, all_states_size, hipMemcpyHostToDevice );
    check_cuda_errors("(hipMemcpy dev_fault_free_states to GPU)");

    Gate* dev_gates;
    cuda_malloc_bytes += (sizeof(Gate) * num_gates);
    hipMalloc( (void**)&dev_gates, sizeof(Gate) * num_gates);
    check_cuda_errors("(hipMalloc dev_gates)");
    assert(dev_gates != NULL);
    hipMemcpy( dev_gates, gates, sizeof(Gate) * num_gates, hipMemcpyHostToDevice );
    check_cuda_errors("(hipMemcpy dev_gates to GPU)");

    // Launch Kernel 1! We have a blocks with FAULTS_PER_BLOCK_KERNEL_1 threads, one thread for each test
    gettimeofday(&tvPreK1, NULL);
    cuda_fault_free_fault_sim<<< divide_round_up(num_tests, FAULTS_PER_BLOCK_KERNEL_1), FAULTS_PER_BLOCK_KERNEL_1 >>>(dev_gates, num_gates, dev_fault_free_states, state_bytes, num_tests);
    check_cuda_errors("1 (fault free fault simulation)");
    gettimeofday(&tvPostK1, NULL);
    printf("finished with fault-free responses kernel #1\n");

    hipMemcpy( fault_free_states, dev_fault_free_states, all_states_size, hipMemcpyDeviceToHost );
    check_cuda_errors("(hipMemcpy dev_fault_free_states to CPU)");
#if 0
    // For TRAX Multi-Fault Injections we want to get an idea of which gates spend their time outputting 1, which increases NBTI (PMOS are turned on to output 1)
    //print_state(fault_free_states, num_state_values);
    //print_state_raw(fault_free_states, num_state_values);

    uint *count_newly_damaged_at_test = (uint*) malloc(sizeof(uint) * num_tests);
    memset(count_newly_damaged_at_test, 0, sizeof(uint) * num_tests);

    uint *net_damaged = (uint*) malloc(sizeof(uint) * num_nets);
    memset(net_damaged, 0, sizeof(uint) * num_nets);

    printf("Analyzing fault-free states to determine usage of %d nets...\n", num_nets);
    uint damage_threshold = (0.7 * 2 * num_tests);
    fp = fopen(filename_usage, "w");
    for (uint net_id = 0; net_id < num_nets; net_id++) {
        uint high_count = 0;
        printf("\r%f", (1 + net_id) / (1.0 * num_nets));
        for (uint test_id = 0; test_id < num_tests; test_id++) {
            uchar *this_state = fault_free_states + test_id * state_bytes;
            // Need to check both v1 and v2 for each test
            // In the fault-free circuit there can only be 0 and 1 values
            if (this_state[net_id * 2] == LOGIC_1) {
                high_count++;
            }
            if (this_state[net_id * 2 + 1] == LOGIC_1) {
                high_count++;
            }
            // Check if this test has caused this net to go newly-damaged
            // Accumulating damage when the gate input is low
            uint low_count = (2 * (test_id + 1)) - high_count;
            if ((!net_damaged[net_id]) && (low_count == damage_threshold)) {
                // We've seen more than 70% low counts, it's damaged
                count_newly_damaged_at_test[test_id] += 1;
                net_damaged[net_id] = 1;
            }
        }
        fprintf(fp, "%d,%f\n", net_id, high_count / (2.0 * num_tests));
    }
    printf("\n");
    fclose(fp);
    printf("Finished saving the usage info...\n");

    fp = fopen("count_newly_damaged_at_test", "w");
    for (uint test_id = 0; test_id < num_tests; test_id++) {
        fprintf(fp, "%d,%d\n", test_id, count_newly_damaged_at_test[test_id]);
    }
    fclose(fp);

    exit(1);
#endif



/*_  ________ _____  _   _ ______ _        ___
 | |/ /  ____|  __ \| \ | |  ____| |      |__ \
 | ' /| |__  | |__) |  \| | |__  | |         ) |
 |  < |  __| |  _  /| . ` |  __| | |        / /
 | . \| |____| | \ \| |\  | |____| |____   / /_
 |_|\_\______|_|  \_\_| \_|______|______| |____|
*/

    // Now, at this point, we have the fault-free responses for all tests
    // From the grand plan: "2. Then, for each fault, we determine which test pairs activate the fault."
    // Each thread corresponds with a single fault, and determines which tests activate the fault
    // NEW PLAN: We need to pack this data tighter using BIT_SET_UCHAR
    uint size_fault_activations = divide_round_up(num_tests * num_gates * 2, 8);
    pretty_bytes(buffer, size_fault_activations);
    printf("We need %s to store %ld potential fault activation bits\n", buffer, ((unsigned long) num_tests) * num_gates * 2);
    malloc_bytes += (size_fault_activations);
    uchar* fault_activations = (uchar*) malloc(size_fault_activations);
    assert (fault_activations != NULL);
    memset(fault_activations, 0, size_fault_activations);

    uchar* dev_fault_activations;
    cuda_malloc_bytes += size_fault_activations;
    hipMalloc( (void**)&dev_fault_activations, size_fault_activations);
    check_cuda_errors("hipMalloc (dev_fault_activations)");
    assert(dev_fault_activations != NULL);
    // The memcpy below just copies in zeros. Is there a way to get around this, maybe an initializing cudaMalloc() ? TODO
    hipMemcpy( dev_fault_activations, fault_activations, size_fault_activations, hipMemcpyHostToDevice );
    check_cuda_errors("hipMemcpy (dev_fault_activations zeros to GPU)");

    // Each thread checks all tests to see which tests activate its fault.
    // dev_fault_free_states is still in the GPU, no need to copy it back and forth between kernels!
    gettimeofday(&tvPreK2, NULL);
    uint num_blocks_kernel_2 = divide_round_up(num_gates * 2, FAULTS_PER_BLOCK_KERNEL_2);
    cuda_check_fault_activations<<< num_blocks_kernel_2, FAULTS_PER_BLOCK_KERNEL_2 >>>(dev_gates, dev_fault_free_states, state_bytes, num_tests, dev_fault_activations, num_gates * 2);
    check_cuda_errors("2 (fault activations)");
    gettimeofday(&tvPostK2, NULL);

    hipMemcpy( fault_activations, dev_fault_activations, size_fault_activations, hipMemcpyDeviceToHost );
    check_cuda_errors("post-2 (hipMemcpy fault_activations to CPU)");


/*_  ________ _____  _   _ ______ _        ____
 | |/ /  ____|  __ \| \ | |  ____| |      |___ \
 | ' /| |__  | |__) |  \| | |__  | |        __) |
 |  < |  __| |  _  /| . ` |  __| | |       |__ <
 | . \| |____| | \ \| |\  | |____| |____   ___) |
 |_|\_\______|_|  \_\_| \_|______|______| |____/

     * 4. We do another parallel fault simulation on only the patterns that
     *    activate the fault. We also have to change each state to put the X
     *    value at the necessary net before fault simulation (sequential?). */
    // At this point each fault has some number of tests that activate the fault.
    // These are the only tests we need to further simulate.
    // NEW PLAN: We'll definitely have > 512 activations for larger circuits, so we have two options:
    // 1. Separate kernel invocations for each fault (THIS IS THE CHOICE I MADE GOING FORWARD)
    //    + Re-use the faulty states memory for each fault (don't need to allocate gigs and gigs up front)
    //    + Can take advantage of "later faults skip most data" speedups
    //    - Kernel overhead of having thousands of kernel invocations
    // 2. One gigantic kernel
    //    - All threads must process entire circuit (so they stay in sync) and lose the "later faults skip most data" speedup
    //    - Need to allocate num_activations * state_bytes bytes of memory (which can be huge)
    //    + Only one kernel invocation, so we avoid any/all overhead with kernel calls
    // As noted above, we decided to go with option 1, which seems to be working well for now.
    // UPDATE MAY 2017 - This was possibly a bad plan. There is a lot of overhead in each kernel invocation. Going to try option 2 now.

    // Need an array of how many tests need to be run (for each fault)
    malloc_bytes += (sizeof(uint) * num_gates * 2);
    uint* num_fault_activations = (uint*) malloc(sizeof(uint) * num_gates * 2);
    assert(num_fault_activations != NULL);

    // This is the array of offsets into the big state table (for each fault)
    malloc_bytes += (sizeof(uint) * num_gates * 2);
    uint* fault_activations_offset = (uint*) malloc(sizeof(uint) * num_gates * 2);
    assert(fault_activations_offset != NULL);

    ulong total_activations = 0;
    ulong max_num_activations = 0;
    for (uint fault_id = 0; fault_id < num_gates * 2; fault_id++)
    {
        fault_activations_offset[fault_id] = total_activations;

        uint count = 0;
        for (uint test_id = 0; test_id < num_tests; test_id++)
        {
            count += BIT_GET_UCHAR(fault_activations, fault_id * num_tests + test_id);
        }
        num_fault_activations[fault_id] = count;
        //printf("Fault %8d activated by %8d tests\n", fault_id, count);

        total_activations += count;
        if (count > max_num_activations)
        {
            max_num_activations = count;
        }
    }
    printf("Max num activations: %ld\n", max_num_activations);
    //printf("------------------------------------------------\n");
    printf("total_activations: %ld\n", total_activations);

    uint size_faulty_states = state_bytes * total_activations;
    pretty_bytes(buffer, size_faulty_states);
    printf("We require %s for our faulty states!\n", buffer);
    malloc_bytes += (size_faulty_states);
    uchar* faulty_states = (uchar*) malloc(size_faulty_states);
    assert(faulty_states != NULL);

    // let's make an array of the test_id values for each fault, in order for fault_0, then fault_1, etc
    // Note, we can't merge this pair of loops with the very similar pair of loops above, because we need to know total_activations before we can malloc here
    //      It's not a big deal because this part doesn't take much of the time
    malloc_bytes += (sizeof(uint) * total_activations);
    uint* activating_test_ids = (uint*) malloc(sizeof(uint) * total_activations);
    assert(activating_test_ids != NULL);
    uint array_index = 0;
    for (uint fault_id = 0; fault_id < num_gates * 2; fault_id++)
    {
        for (uint test_id = 0; test_id < num_tests; test_id++)
        {
            if (BIT_GET_UCHAR(fault_activations, fault_id * num_tests + test_id))
            {
                activating_test_ids[array_index] = test_id;

                // Copy corresponding fault-free test row into the faulty_states array
                uchar* fault_free_state = fault_free_states + state_bytes * test_id;
                uchar* faulty_state = faulty_states + state_bytes * array_index;
                memcpy(faulty_state, fault_free_state, state_bytes);
                // Activate fault in faulty_states
                uint my_gate_id = fault_id / 2;
                // 0x80 = "activated fault" so we avoid re-evaluating the gate and losing the activated fault!
                faulty_state[my_gate_id * 2 + 1] = 0x80 |
                #if USE_TRAX
                    LOGIC_X; // Activate the fault by marking an X at the fault site in v2
                #else
                    faulty_state[my_gate_id * 2]; // Activate the fault by copying-in the v1 value (infinitely delayed transition)
                #endif

                array_index += 1;
            }
        }
    }

    // May 2017 PLAN - There's too much overhead with each kernel 3 invocation, so let's try to do one gigantic parallel simulation of all the activated faults.
    // We copy fault-free test rows from the fault-free circuit stats array based on the activating_test_ids array above, then activate the faults,
    // then simulate the circuits in parallel, then copy back the faulty circuit stats, and finally generate the dictionary data.
    uchar* dev_faulty_states;
    cuda_malloc_bytes += size_faulty_states;
    hipMalloc( (void**)&dev_faulty_states, size_faulty_states);
    check_cuda_errors("pre-3 (hipMalloc faulty_states)");
    assert(dev_faulty_states != NULL);

    uint dict_size = divide_round_up(num_faults * num_tests, 8);
    pretty_bytes(buffer, dict_size);
    printf("We require %s for the packed dictionary data (%d faults, %d tests)\n", buffer, num_faults, num_tests);
    malloc_bytes += dict_size;
    uchar* dict = (uchar*) malloc(dict_size);
    assert(dict != NULL);
    memset(dict, 0, dict_size);

    pretty_bytes(buffer, malloc_bytes);
    printf("malloc_bytes: %ld (%s)\n", malloc_bytes, buffer);
    pretty_bytes(buffer, cuda_malloc_bytes);
    printf("cuda_malloc_bytes: %ld (%s)\n", cuda_malloc_bytes, buffer);

    struct timeval tvStep;
    gettimeofday(&tvPreK3, NULL);
    gettimeofday(&tvStep, NULL);
#define USE_CUDA_PROFILER (0)
#if USE_CUDA_PROFILER
    cudaProfilerStart();
#endif
    // Copy the activated faulty states from cpu to gpu
    hipMemcpy(dev_faulty_states, faulty_states, size_faulty_states, hipMemcpyHostToDevice);
    check_cuda_errors("pre-3 (hipMemcpy faulty_states -> dev_faulty_states - CPU-to-GPU)");

    // Run kernel 3
    hipDeviceSynchronize(); //TODO what's this for?
    uint num_blocks = divide_round_up(total_activations, FAULTS_PER_BLOCK_KERNEL_3);
    cuda_faulty_fault_sim<<< num_blocks, FAULTS_PER_BLOCK_KERNEL_3 >>>(dev_gates, num_gates, dev_faulty_states, state_bytes, total_activations);
    check_cuda_errors("3 (faulty fault sim)");
    hipDeviceSynchronize(); //TODO what's this for?

    // Copy the faulty states from gpu to cpu
    hipMemcpy(faulty_states, dev_faulty_states, size_faulty_states, hipMemcpyDeviceToHost);
    check_cuda_errors("post-3 (hipMemcpy dev_faulty_states -> faulty_states - GPU-to-CPU)");

#if USE_CUDA_PROFILER
    cudaProfilerStop();
#endif
    gettimeofday(&tvPostK3, NULL);
    printf("\n");

    // Analyze the faulty states to generate the pass/fail dictionary data
    array_index = 0;
    for (uint fault_id = 0; fault_id < num_gates * 2; fault_id++)
    {
        for (uint test_id = 0; test_id < num_tests; test_id++)
        {
            if (BIT_GET_UCHAR(fault_activations, fault_id * num_tests + test_id))
            {
                uchar* faulty_state = faulty_states + state_bytes * array_index;
                if (faulty_state[state_bytes - 1]) {
                    // fail
                    BIT_SET_UCHAR(dict, fault_id * num_tests + test_id, 1);
                }

                array_index++;
            }
        }
    }

    // Now, write the dictionary data to disk.
    fp = fopen(filename_dictionary, "w");
    for (uint fault_list_index = 0; fault_list_index < num_faults; fault_list_index++)
    {
        for (uint test_id = 0; test_id < num_tests; test_id++)
        {
            fprintf(fp, "%d", BIT_GET_UCHAR(dict, fault_list_index * num_tests + test_id));
        }
        fprintf(fp, "\n");
    }
    fclose(fp);
    gettimeofday(&tvEnd, NULL);

    //printf("----------------------------------\n");
    printf("Wrote dictionary to '%s', goodbye!\n", filename_dictionary);
    printf("----------------------------------\n");
    printf("Detailed timing information:\n");

    timeval_subtract(&tvDiff, &tvDoneParsing, &tvStart);
    printf("1. %ld.%06ld Parse the input files: \n", tvDiff.tv_sec, tvDiff.tv_usec);

    timeval_subtract(&tvDiff, &tvPreK1, &tvDoneParsing);
    printf("2. %ld.%06ld Get ready for kernel 1\n", tvDiff.tv_sec, tvDiff.tv_usec);

    timeval_subtract(&tvDiff, &tvPostK1, &tvPreK1);
    printf("3. %ld.%06ld Kernel 1\n", tvDiff.tv_sec, tvDiff.tv_usec);

    timeval_subtract(&tvDiff, &tvPreK2, &tvPostK1);
    printf("4. %ld.%06ld Get ready for kernel 2\n", tvDiff.tv_sec, tvDiff.tv_usec);

    timeval_subtract(&tvDiff, &tvPostK2, &tvPreK2);
    printf("5. %ld.%06ld Kernel 2\n", tvDiff.tv_sec, tvDiff.tv_usec);

    timeval_subtract(&tvDiff, &tvPreK3, &tvPostK2);
    printf("6. %ld.%06ld Get ready for kernel 3\n", tvDiff.tv_sec, tvDiff.tv_usec);

    timeval_subtract(&tvDiff, &tvPostK3, &tvPreK3);
    printf("7. %ld.%06ld Kernel 3 loop\n", tvDiff.tv_sec, tvDiff.tv_usec);

    timeval_subtract(&tvDiff, &tvEnd, &tvPostK3);
    printf("8. %ld.%06ld Writing dictionary to file\n", tvDiff.tv_sec, tvDiff.tv_usec);

    printf("Total time:\n");
    timeval_subtract(&tvDiff, &tvEnd, &tvStart);
    printf("   %ld.%06ld\n", tvDiff.tv_sec, tvDiff.tv_usec);


    // free up the allocated CUDA memory
    hipFree(dev_fault_free_states);
    hipFree(dev_gates);
    hipFree(dev_fault_activations);
    hipFree(dev_faulty_states);

    // free up the allocated CPU memory
    // TODO double-check all these CPU-side free calls again!
    free(filename_input);
    free(filename_tests);
    free(filename_dictionary);
    free(filename_faults);
    free(inputs);
    free(outputs);
    free(gates);
    free(tests_v1);
    free(tests_v2);
    free(tests_expected);
    free(faults);

    free(fault_free_states);
    free(fault_activations);
    free(num_fault_activations);
    free(fault_activations_offset);
    free(activating_test_ids);
    free(faulty_states);
    free(dict);

    // Apparently some profiling data is transfered asyncronously so we have to call this function to wait for those transfers to finish
    hipDeviceReset();

    return 0;
}

