/*

   GPU fault simulator for TRAX fault model
   Matthew Beckler
   18-645 How to Write Fast Code
   Carnegie Mellon University
   Fall 2012

   Last Updated: August 1, 2013
   Copyright (c) 2013, Matthew Beckler

   This program is free software; you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation; either version 2 of the License, or
   (at your option) any later version.

   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License along
   with this program; if not, write to the Free Software Foundation, Inc.,
   51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.


   Usage: ./fault_sim basename
   Looks for the files basename/basename.easy and basename/basename.tests and TODO
   Stores dictionary output in dictionary.cuda

   For more information about the TRAX fault model:
   Beckler, M. and Blanton, R.D., "On-Chip Diagnosis for Early-Life and Wear-Out Failures," IEEE International Test Conference, Nov. 2012.

   TODO list of things to improve:

   * Figure out what's going on with dev_gates, didn't we put this in constants memory?
        Nope, only the gate eval LuT is in constants memory. There's only 64k of constants mem, and we need 128 bytes for the gate eval LuT. Each Gate takes up 12 bytes (and we could only reduce that by limiting us to 2^16 = 65536 nets which might not be ok) so we can have a maximum of 5450 gates, which is not enough. Unless we could use multiple kernels for each chunk of 5450 gates? Probably not worth it.

   * Maybe use shared memory for the netlist information?
        Probably not enough room for the larger circuits we want to handle

   * Store dictionary in memory/disk as binary format?

   * Some way to do multiple gate evals per thread? Maybe a packed data structure? Maybe machine word-width algebraic operations like in that one DAC submission?

   * As mentioned above, some data storage techniques had to be migrated to a compacted storage technique, where multiple 1 or 2 bit values are stored in a single byte. This reduces the total storage required, but makes each access more complex, usually involving one or more bit shift and bitwise masking operations. It would be good to analyze the actual access patterns for these memories, and determine if functions such as "extract four consecutive values" or "extract both v1 and v2 for a given net" would be useful and faster than individual accesses. Additionally, for smaller circuits or on systems with more available memory, it would be useful to be able to automatically detect the memory size and switch between data storage formats based on the system details.

   * There are a number of small optimizations we would like to investigate with regards to array-of-struct vs struct-of-arrays, especially in our gate and test memory storage.

   Update log:
   * Added support for TF faults in addition to just TRAX faults - Aug 2013

*/

#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <math.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

/* Here is the "easy" to parse file format for the circuit netlist:
    NUMINPUTS 5
    INPUT 8
    INPUT 9
    INPUT 10
    INPUT 11
    INPUT 12
    NUMOUTPUTS 4
    OUTPUT 6
    OUTPUT 7
    OUTPUT 1
    OUTPUT 0
    NUMGATES 8
    1 0 9 9
    0 1 8 8
    2 2 8 10
    3 3 10 11
    4 4 9 3
    5 5 3 12
    8 6 2 4
    9 7 4 5

    Nets are integers.
    Gates can only have two inputs (buf/inv should have the same net listed as both inputs, the second will be ignored, but won't break the parser).
    GATES ABSOLUTELY MUST BE TOPOLOGICALLY SORTED based on their level within the circuit.
    Gates' output net ids must start at 0.
    There are probably other implicit assumptions that I have forgotten about. Caveat emptor
*/

#define uchar unsigned char
#define uint unsigned int

// This is the definition of all eight supported gate types. Ordering is to make the GPU code more efficient (and/nand/or/nor are commonly grouped together, same with xor/xnor)
#define TYPE_AND    (0)
#define TYPE_NAND   (1)
#define TYPE_OR     (2)
#define TYPE_NOR    (3)
#define TYPE_BUF    (4)
#define TYPE_INV    (5)
#define TYPE_XOR    (6)
#define TYPE_XNOR   (7)

// Since we have a four-valued logic, we need two bits to represent each one.
// I was originally trying to be creative about these assignments, to make it so that we could do gate evaluations in parallel using bit-operation instructions.
// Which is how I decided to make 0 = 00, 1 = 11, and X and H be the other two.
// However, since we haven't implemented parallel gate evaluations, this is kind of a moot point. However these values are indeed being used.
#define LOGIC_0 (0)
#define LOGIC_X (1)
#define LOGIC_H (2)
#define LOGIC_1 (3)

// How many threads per block? Maximum is 512, make sure each is a multiple of 16, multiple of 32 is probably better.
// Doesn't really seem to affect performance much, so something else is probably limiting the speed.
#define FAULTS_PER_BLOCK_KERNEL_1 (512)
#define FAULTS_PER_BLOCK_KERNEL_2 (512)
#define FAULTS_PER_BLOCK_KERNEL_3 (512)

// FYI, pointers cost 8 bytes!

// Structure to represent a gate
// We should pack in1 and in2 together in the same int if we need to save memory
typedef struct
{
    uchar type;
    uchar is_output;
    uint in1;
    uint in2;
} Gate; // 12 bytes due to nice packing

// Structure to represent a fault
typedef struct
{
    uint net;
    uint polarity; // 0 = falling, 1 = rising
} Fault;


// Used to evaluate how long (wall-clock) the three kernels take to run. Taken from:
// http://stackoverflow.com/questions/1468596/c-programming-calculate-elapsed-time-in-milliseconds-unix
int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1)
{
    long int diff = (t2->tv_usec + 1000000 * t2->tv_sec) - (t1->tv_usec + 1000000 * t1->tv_sec);
    result->tv_sec = diff / 1000000;
    result->tv_usec = diff % 1000000;

    return (diff<0);
}
// Accumulates (end - start) into result
void timeval_accumulate_diff(struct timeval *result, struct timeval *start, struct timeval *end)
{
    long int diff_usec = (end->tv_usec + 1000000 * end->tv_sec) - (start->tv_usec + 1000000 * start->tv_sec);
    diff_usec += result->tv_usec;
    result->tv_sec += diff_usec / 1000000;
    result->tv_usec = diff_usec % 1000000;
}

// Prints a nice number of bytes (KB, MB, GB, etc) to the provided buffer.
void pretty_bytes(char* buf, uint bytes)
{
    const char* suffixes[] = {"B", "KB", "MB", "GB", "TB", "PB", "EB"};
    uint order = 0;
    double count = bytes;
    while ((count >= 1024) && (order < sizeof(suffixes))) {
        order++;
        count /= 1024;
    }
    if ((count - floor(count)) == 0.0) {
        sprintf(buf, "%d %s", (int)count, suffixes[order]);
    } else {
        sprintf(buf, "%.1f %s", count, suffixes[order]);
    }
}


char printable_logic_value(char value)
{
    switch (value)
    {
        case LOGIC_0: return '0';
        case LOGIC_X: return 'X';
        case LOGIC_H: return 'H';
        case LOGIC_1: return '1';
    }
    return '?';
}


// Circuit states are stored as pairs of values (v1, v2) for each net in the circuit.
void print_state(uchar* state, uint num_state_values)
{
    for (uint i = 0; i < num_state_values; i++)
    {
        uchar s = state[i];
        printf("%c", printable_logic_value(s));
        if (i % 2 == 1)
            printf(" ");
    }
    printf("\n");
}
void print_state_raw(uchar* state, uint num_state_values)
{
    for (uint i = 0; i < num_state_values; i++)
    {
        printf("%X", state[i]);
    }
    printf("\n");
}

// For some data like the tests values we pack 8 1-bit values into each byte.
// These are the accessor functions to make it easy to access that data.
__host__ __device__ void BIT_SET_UCHAR(uchar* array, uint which, uchar value)
{
    uchar* b = array + (which / 8);
    uchar shift = which % 8;
    if (value)
        *b |= (1 << shift);
    else
        *b &= ~(1 << shift);
}
__host__ __device__ uchar BIT_GET_UCHAR(uchar* array, uint which)
{
    uchar* b = array + (which / 8);
    uchar shift = which % 8;
    if (*b & (1 << shift))
        return 1;
    else
        return 0;
}

// This is the new gate evaluation lookup table.
// Simply concatenate the gate type with the four bits of input values.
__constant__ uchar gate_eval_lut[128] = {
    // gate 000 (and)
    LOGIC_0, // 0 0 -> 0
    LOGIC_0, // 0 X -> 0
    LOGIC_0, // 0 H -> 0
    LOGIC_0, // 0 1 -> 0
    LOGIC_0, // X 0 -> 0
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X
    LOGIC_X, // X 1 -> X
    LOGIC_0, // H 0 -> 0
    LOGIC_X, // H X -> X
    LOGIC_H, // H H -> H
    LOGIC_H, // H 1 -> H
    LOGIC_0, // 1 0 -> 0
    LOGIC_X, // 1 X -> X
    LOGIC_H, // 1 H -> H
    LOGIC_1, // 1 1 -> 1

    // gate 001 (nand)
    LOGIC_1, // 0 0 -> 1
    LOGIC_1, // 0 X -> 1
    LOGIC_1, // 0 H -> 1
    LOGIC_1, // 0 1 -> 1
    LOGIC_1, // X 0 -> 1
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X
    LOGIC_X, // X 1 -> X
    LOGIC_1, // H 0 -> 1
    LOGIC_X, // H X -> X
    LOGIC_H, // H H -> H
    LOGIC_H, // H 1 -> H
    LOGIC_1, // 1 0 -> 1
    LOGIC_X, // 1 X -> X
    LOGIC_H, // 1 H -> H
    LOGIC_0, // 1 1 -> 0

    // gate 010 (or)
    LOGIC_0, // 0 0 -> 0
    LOGIC_X, // 0 X -> X
    LOGIC_H, // 0 H -> H
    LOGIC_1, // 0 1 -> 1
    LOGIC_X, // X 0 -> X
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X
    LOGIC_1, // X 1 -> 1
    LOGIC_H, // H 0 -> H
    LOGIC_X, // H X -> X
    LOGIC_H, // H H -> H
    LOGIC_1, // H 1 -> 1
    LOGIC_1, // 1 0 -> 1
    LOGIC_1, // 1 X -> 1
    LOGIC_1, // 1 H -> 1
    LOGIC_1, // 1 1 -> 1

    // gate 011 (nor)
    LOGIC_1, // 0 0 -> 1
    LOGIC_X, // 0 X -> X
    LOGIC_H, // 0 H -> H
    LOGIC_0, // 0 1 -> 0
    LOGIC_X, // X 0 -> X
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X
    LOGIC_0, // X 1 -> 0
    LOGIC_H, // H 0 -> H
    LOGIC_X, // H X -> X
    LOGIC_H, // H H -> H
    LOGIC_0, // H 1 -> 0
    LOGIC_0, // 1 0 -> 0
    LOGIC_0, // 1 X -> 0
    LOGIC_0, // 1 H -> 0
    LOGIC_0, // 1 1 -> 0

    // gate 100 (buffer)
    LOGIC_0, // 0 0 -> 0
    LOGIC_0, // 0 X -> 0 - not possible, for BUF/INV must have same values
    LOGIC_0, // 0 H -> 0 - not possible, for BUF/INV must have same values
    LOGIC_0, // 0 1 -> 0 - not possible, for BUF/INV must have same values
    LOGIC_X, // X 0 -> X - not possible, for BUF/INV must have same values
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X - not possible, for BUF/INV must have same values
    LOGIC_X, // X 1 -> X - not possible, for BUF/INV must have same values
    LOGIC_H, // H 0 -> H - not possible, for BUF/INV must have same values
    LOGIC_H, // H X -> H - not possible, for BUF/INV must have same values
    LOGIC_H, // H H -> H
    LOGIC_H, // H 1 -> H - not possible, for BUF/INV must have same values
    LOGIC_1, // 1 0 -> H - not possible, for BUF/INV must have same values
    LOGIC_1, // 1 X -> H - not possible, for BUF/INV must have same values
    LOGIC_1, // 1 H -> H - not possible, for BUF/INV must have same values
    LOGIC_1, // 1 1 -> H

    // gate 001 (inverter)
    LOGIC_1, // 0 0 -> 1
    LOGIC_1, // 0 X -> 1 - not possible, for BUF/INV must have same values
    LOGIC_1, // 0 H -> 1 - not possible, for BUF/INV must have same values
    LOGIC_1, // 0 1 -> 1 - not possible, for BUF/INV must have same values
    LOGIC_X, // X 0 -> X - not possible, for BUF/INV must have same values
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X - not possible, for BUF/INV must have same values
    LOGIC_X, // X 1 -> X - not possible, for BUF/INV must have same values
    LOGIC_H, // H 0 -> H - not possible, for BUF/INV must have same values
    LOGIC_H, // H X -> H - not possible, for BUF/INV must have same values
    LOGIC_H, // H H -> H
    LOGIC_H, // H 1 -> H - not possible, for BUF/INV must have same values
    LOGIC_0, // 1 0 -> 0 - not possible, for BUF/INV must have same values
    LOGIC_0, // 1 X -> 0 - not possible, for BUF/INV must have same values
    LOGIC_0, // 1 H -> 0 - not possible, for BUF/INV must have same values
    LOGIC_0, // 1 1 -> 0

    // gate 110 (xor)
    LOGIC_0, // 0 0 -> 0
    LOGIC_X, // 0 X -> X
    LOGIC_H, // 0 H -> H
    LOGIC_1, // 0 1 -> 1
    LOGIC_X, // X 0 -> X
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X
    LOGIC_X, // X 1 -> X
    LOGIC_H, // H 0 -> H
    LOGIC_X, // H X -> X
    LOGIC_H, // H H -> H
    LOGIC_H, // H 1 -> H
    LOGIC_1, // 1 0 -> 1
    LOGIC_X, // 1 X -> X
    LOGIC_H, // 1 H -> H
    LOGIC_0, // 1 1 -> 0

    // gate 111 (xnor)
    LOGIC_1, // 0 0 -> 1
    LOGIC_X, // 0 X -> X
    LOGIC_H, // 0 H -> H
    LOGIC_0, // 0 1 -> 0
    LOGIC_X, // X 0 -> X
    LOGIC_X, // X X -> X
    LOGIC_X, // X H -> X
    LOGIC_X, // X 1 -> X
    LOGIC_H, // H 0 -> H
    LOGIC_X, // H X -> X
    LOGIC_H, // H H -> H
    LOGIC_H, // H 1 -> H
    LOGIC_0, // 1 0 -> 0
    LOGIC_X, // 1 X -> X
    LOGIC_H, // 1 H -> H
    LOGIC_1, // 1 1 -> 1
};


// This is the core cuda fault simulation function, only called from the two fault sim wrapper functions (kernels 1 and 3).
// It will probably be inlined into each kernel by the compiler, but this is better for future maintenance
// Inputs:
//      gates           The global, unchanging gate structure
//      my_state        Pointer to the state for this simulation (explained above, basically it's just net0_v1, net0_v2, net1_v1, net1_v2, etc)
// Outputs:
//      Updates my_state in place with the correct values after fault simulation
// Returns:
//      The v2 value
#define TRAN0110 ( (LOGIC_0 << 6) | (LOGIC_1 << 4) | (LOGIC_1 << 2) | LOGIC_0 )
#define TRAN1001 ( (LOGIC_1 << 6) | (LOGIC_0 << 4) | (LOGIC_0 << 2) | LOGIC_1 )
#define TRAN01H0 ( (LOGIC_0 << 6) | (LOGIC_1 << 4) | (LOGIC_H << 2) | LOGIC_0 )
#define TRAN100H ( (LOGIC_1 << 6) | (LOGIC_0 << 4) | (LOGIC_0 << 2) | LOGIC_H )
#define TRAN011H ( (LOGIC_0 << 6) | (LOGIC_1 << 4) | (LOGIC_1 << 2) | LOGIC_H )
#define TRAN10H1 ( (LOGIC_1 << 6) | (LOGIC_0 << 4) | (LOGIC_H << 2) | LOGIC_1 )
__device__ uchar cuda_fault_sim_core(const Gate* g, uchar* my_state, uint gate_id, uchar use_trax)
{
    // could we transpose our state matrix to make the accesses align better? TODO
    // Each thread is accessing the same net at the same time, maybe we could make those accesses be coallesced?
    uchar in1_v1 = my_state[g->in1 * 2];
    uchar in2_v1 = my_state[g->in2 * 2];
    uchar in1_v2 = my_state[g->in1 * 2 + 1];
    uchar in2_v2 = my_state[g->in2 * 2 + 1];

    // Using the new gate evaluation lookup table, we use the gate type and two input values to craft a 7-bit index
    uchar v1 = gate_eval_lut[(g->type << 4) | (in1_v1 << 2) | (in2_v1)];
    uchar v2 = gate_eval_lut[(g->type << 4) | (in1_v2 << 2) | (in2_v2)];

    if (use_trax)
    {
        // Now we need to detect if v2 should result in a hazard due to this gate's inputs
        uchar merged_values = (in1_v1 << 6) | (in2_v1 << 4) | (in1_v2 << 2) | in2_v2;
        // and/nand = 0/1, or/nor = 2/3
        uchar hazard_and_nand = (g->type <= 1) &&                 (merged_values == TRAN0110 || merged_values == TRAN1001 || merged_values == TRAN01H0 || merged_values == TRAN100H);
        uchar hazard_or_nor   = (g->type <= 3 && g->type >= 2) && (merged_values == TRAN0110 || merged_values == TRAN1001 || merged_values == TRAN011H || merged_values == TRAN10H1);
        uchar hazard_xor_xnor = (g->type >  5) && (v1 == v2 && in1_v1 != in1_v2 && in2_v1 != in2_v2);
        uchar hazard = hazard_and_nand || hazard_or_nor || hazard_xor_xnor;
        v2 = hazard ? LOGIC_H : v2;
    }

    // Update my_state in place with the (potentially new) values of v1 and v2:
    // Note that unlike the reference implementation, we don't care if the values were updated, since we check all downstream gates regardless.
    my_state[gate_id * 2] = v1;
    my_state[gate_id * 2 + 1] = v2;

    return v2;
}


// This is Kernel 1, the cuda fault-free fault simulation function. Pass in the gates, all the states (one per test), the state size, and the number of gates.
// Updates all the states in place. One thread per test pair, FAULTS_PER_BLOCK_KERNEL_1 threads per block.
__global__ void cuda_fault_free_fault_sim(Gate* gates, uint num_gates, uchar* all_states, uint state_bytes, uint num_tests, uchar use_trax)
{
    // each thread will go through a complete fault simulation
    // since we are not skipping around, all threads stay in lock-step
    // Key idea: branching is ok as long as all the threads do the same thing
    // This means we can't have any data-dependent branching
    // Since each thread evaluates the same gate at the same time, that kind of branching is ok
    // However we can't have branching based on the actual logic values going through the gate

    uint test_id = (blockIdx.x * FAULTS_PER_BLOCK_KERNEL_1) + threadIdx.x;
    if (test_id < num_tests)
    {
        uchar* my_state = all_states + (state_bytes * test_id);
        // We iterate over all the relevant gates, in topological order (this sorting was already handled, our netlist comes pre-sorted)
        for (uint gate_id = 0; gate_id < num_gates; gate_id++)
        {
            cuda_fault_sim_core(&gates[gate_id], my_state, gate_id, use_trax);
        }
    }
}

// This is Kernel 2, which determines which tests activate each fault. One thread per fault.
// It stores a 0 or 1 into the matrix fault_activation, which now uses a packed format (one bit for every fault-test pair), each row is a fault, each col is a test. (initialized to all zeros).
__global__ void cuda_check_fault_activations(Gate* gates, uchar* all_states, uint state_bytes, uint num_tests, uchar* fault_activations, uint num_faults)
{
    uint fault_id = blockIdx.x * FAULTS_PER_BLOCK_KERNEL_2 + threadIdx.x;
    if (fault_id < num_faults)
    {
        uint gate_id = fault_id / 2;
        uint rising = (fault_id % 2); // rising when fault_id is odd
        // this thread has to check for fault activation for fault_id for all tests
        for (uint test_id = 0; test_id < num_tests; test_id++)
        {
            uchar* my_state = all_states + test_id * state_bytes;
            uchar v1 = my_state[gate_id * 2];
            uchar v2 = my_state[gate_id * 2 + 1];
            uchar activated = ( ( rising && (v1 == LOGIC_0) && (v2 == LOGIC_1)) || // output transition activation
                                (!rising && (v1 == LOGIC_1) && (v2 == LOGIC_0)) || // ditto
                                (v2 == LOGIC_H) );                                 // hazard-based activation - And actually, if we are not doing TRAX fault sim, then there can be no LOGIC_H values, so this activation condition is harmless, neat!
            // argh, this is also susceptible to the problem of multiple threads writing to the same byte concurrently
            //BIT_SET_UCHAR(fault_activations, fault_id * num_tests + test_id, (activated ? 1 : 0));
            // (TODO - This comment is talking about BIT_SET_UCHAR, right? Not the atomicOr, right?)

            uint index = fault_id * num_tests + test_id;
            atomicOr( ((uint*)fault_activations) + (index / 32), activated << (index % 32));
        }
    }
}


// This is Kernel 3, which does the faulty fault simulation. One grid of blocks per fault, FAULTS_PER_BLOCK_KERNEL_3 threads per block, one thread per activating test.
// NEW PLAN: Update all the states in place, but never copy the faulty states back to the CPU.
// Instead, we now have the dictionary row in memory and we directly write the pass/fail bit into that memory directly, and then copy it back to the CPU and write it to disk.
__global__ void cuda_faulty_fault_sim(Gate* gates, uint num_gates,
                                      uchar* faulty_states, uint state_bytes,
                                      uint my_num_fault_activations, uint my_activations_offset, uint* activating_test_ids,
                                      uchar* dict, uint fault_list_index, uint num_tests,
                                      uint fault_id, uchar use_trax)
{
    uint test_offset = blockIdx.x * FAULTS_PER_BLOCK_KERNEL_3 + threadIdx.x;
    if (test_offset < my_num_fault_activations)
    {
        uint test_id = activating_test_ids[my_activations_offset + test_offset];
        uchar* my_state = faulty_states + state_bytes * test_id;
        uint my_gate_id = fault_id / 2; // We only have to start simulating at the fault site, due to the gate ordering!

        if (use_trax) {
            my_state[my_gate_id * 2 + 1] = LOGIC_X; // Activate the fault by marking an X at the fault site in v2
        } else {
            my_state[my_gate_id * 2 + 1] = my_state[my_gate_id * 2]; // Activate the fault by copying-in the v1 value (infinitely delayed transition)
        }

        uchar test_failed = (gates[my_gate_id].is_output ? 1 : 0); // local copy since we'll be writing it many times, copy it to dict[fault_id * num_tests + test_id] eventually.

        // We iterate over all the relevant gates, in topological order (this sorting was already handled, our netlist comes pre-sorted)
        for (uint gate_id = my_gate_id + 1; gate_id < num_gates; gate_id++) // Oh duh, start with the next gate, don't re-evaluate the gate where we just activated a fault (since it will un-activate it!)
        {
            const Gate g = gates[gate_id]; // should be the same for all threads in the kernel
            uchar fault_free_value = my_state[gate_id * 2 + 1];
            uchar v2 = cuda_fault_sim_core(&g, my_state, gate_id, use_trax);

            if (use_trax)
            {
                test_failed = (g.is_output && v2 == LOGIC_X) ? 1 : test_failed; // if an X reaches an output, the test fails
            }
            else
            {
                test_failed = (g.is_output && v2 != fault_free_value) ? 1 : test_failed; // if an output does not match the expected value, the test fails
            }
        }
        uint index = (fault_list_index * num_tests) + test_id;
        atomicOr( ((uint*)dict) + (index / 32), test_failed << (index % 32)); // TODO make this much better or something, cripes
    }
}

void check_cuda_errors(char* kernel_name)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA error in kernel %s: %s\n", kernel_name, hipGetErrorString(err));
        exit(1);
    }
}

// does an integer division of n/d, rounding up
uint divide_round_up(uint n, uint d)
{
    return (n + (d - 1)) / d;
}

// TODO use compile-time directives for the TRAX vs TF code to improve performance?

// This is our main function.
// It would be stellar to split the parsing code into a separate function in a separate file.
int main(int argc, char* argv[])
{
    char buffer[100];  // for using pretty_bytes(buffer, numbytes);

    // I added some timing instrumentation in the code for different sections
    struct timeval tvStart, tvDoneParsing, tvPreK1, tvPostK1, tvPreK2, tvPostK2, tvPreK3, tvPostK3, tvEnd, tvDiff;
    gettimeofday(&tvStart, NULL);

    if (argc != 3)
    {
        fprintf(stderr, "Usage: %s basename {trax|tf}\n", argv[0]);
        exit(1);
    }

    char* basename = argv[1]; // something like "c432"
    uchar use_trax = ((strncmp("trax", argv[2], 5)) == 0);

    char* filename_input = (char*) malloc(2 * strlen(basename) + 6 + 1); // "c432/c432.v", so 2*N + 6 for "/.easy" + 1 for the '\0'
    assert(filename_input != NULL);
    sprintf(filename_input, "%s/%s.easy", basename, basename);
    printf("Netlist filename: '%s'\n", filename_input);

    char* filename_tests = (char*) malloc(2 * strlen(basename) + 12 + 1); // "c432/c432.tests.easy", so 2*N + 12 for "/.tests.easy" + 1 for the '\0'
    assert(filename_tests != NULL);
    sprintf(filename_tests, "%s/%s.tests.easy", basename, basename);
    printf("Tests filename: '%s'\n", filename_tests);

    char* filename_dictionary = (char*) malloc(2 * strlen(basename) + 25 + 1); // "c432/c432.dictionary.trax.pf.cuda", so 2*N + 25 for "/.dictionary.trax.pf.cuda" + 1 for the '\0'
    assert(filename_dictionary != NULL);
    if (use_trax)
    {
        sprintf(filename_dictionary, "%s/%s.dictionary.trax.pf.cuda", basename, basename);
    }
    else
    {
        sprintf(filename_dictionary, "%s/%s.dictionary.tf.pf.cuda", basename, basename);
    }
    printf("Dictionary filename: '%s'\n", filename_dictionary);

    char* filename_faults = (char*) malloc(2 * strlen(basename) + 13 + 1); // "c432/c432.faults.gpu", so 2*N + 13 for "/.faults.gpu" + 1 for the '\0'
    assert(filename_faults != NULL);
    sprintf(filename_faults, "%s/%s.faults.gpu", basename, basename);
    printf("Faults filename: '%s'\n", filename_faults);

    // Circuit netlist data
    uint num_inputs, num_outputs, num_gates;
    uint* inputs;
    uint* outputs;
    Gate* gates;

    // Test patterns data
    uint num_tests;
    uchar *tests_v1;
    uchar *tests_v2;
    uchar *tests_expected;

    // Fault data
    uint num_faults;
    Fault* faults;

    // Ideally, we could just now say something like:
    // read_netlist(&num_inputs, &num_outputs, &num_gates, &num_faults, &inputs, &outputs, &gates);
    // read_tests(&num_tests, &tests);
    // ------------- Begin "please move to separate function/file" section --------------

    // now we need to load in our circuit netlist
    FILE* fp = fopen(filename_input, "r");
    assert(fp != NULL);

    if (fscanf(fp, "NUMINPUTS %d\n", &num_inputs) != 1)
    {
        fprintf(stderr, "Unable to parse NUMINPUTS line!\n");
        exit(1);
    }

    printf("Detected %d inputs\n", num_inputs);
    inputs = (uint*) malloc(sizeof(uint) * num_inputs);
    assert(inputs != NULL);
    for (uint i = 0; i < num_inputs; i++)
    {
        if(fscanf(fp, "INPUT %d\n", &inputs[i]) != 1)
        {
            fprintf(stderr, "Unable to parse INPUT line #%d!\n", i);
            exit(1);
        }
    }


    if (fscanf(fp, "NUMOUTPUTS %d\n", &num_outputs) != 1)
    {
        fprintf(stderr, "Unable to parse NUMOUTPUTS line!\n");
        exit(1);
    }

    printf("Detected %d outputs\n", num_outputs);
    outputs = (uint*) malloc(sizeof(uint) * num_outputs);
    assert(outputs != NULL);
    for (uint i = 0; i < num_outputs; i++)
    {
        if (fscanf(fp, "OUTPUT %d\n", &outputs[i]) != 1)
        {
            fprintf(stderr, "Unable to parse OUTPUT line #%d!\n", i);
            exit(1);
        }
    }


    if (fscanf(fp, "NUMGATES %d\n", &num_gates) != 1)
    {
        fprintf(stderr, "Unable to parse NUMGATES line!\n");
        exit(1);
    }

    // IMPORTANT - We assume that the gates in the file are already in topological order!
    pretty_bytes(buffer, sizeof(Gate) * num_gates);
    printf("Detected %d gates (%s)\n", num_gates, buffer);
    gates = (Gate*) malloc(sizeof(Gate) * num_gates);
    uint type, out, in1, in2;
    for (uint i = 0; i < num_gates; i++)
    {

        if (fscanf(fp, "%d %d %d %d\n", &type, &out, &in1, &in2) != 4)
        {
            fprintf(stderr, "Unable to parse gate line #%d!\n", i);
            exit(1);
        }
        // there has to be a better way to do "if i in list of outputs"
        uchar is_output = 0;
        for (uint output_id = 0; output_id < num_outputs; output_id++)
        {
            if (i == outputs[output_id])
            {
                is_output = 1;
                break;
            }
        }
        gates[i].type = type;
        gates[i].is_output = is_output;
        gates[i].in1 = in1;
        gates[i].in2 = in2;
    }
    fclose(fp);


    // Read in tests
    fp = fopen(filename_tests, "r");
    assert(fp != NULL);

    if (fscanf(fp, "NUMTESTS %d\n", &num_tests) != 1)
    {
        fprintf(stderr, "Unable to parse NUMTESTS line!\n");
        exit(1);
    }

    uint size_v1_v2 = divide_round_up(num_inputs, 8);
    uint size_expected = divide_round_up(num_outputs, 8);
    pretty_bytes(buffer, (size_v1_v2 * 2 + size_expected) * num_tests);
    printf("Detected %d tests (%s)\n", num_tests, buffer);

    // NEW PLAN - Information stored in a compacted format, eight bits per byte, no TestPair structure, just big arrays for v1, v2, and expected, since pointers cost us 8 bytes!
    tests_v1 = (uchar*) malloc(size_v1_v2 * num_tests);
    tests_v2 = (uchar*) malloc(size_v1_v2 * num_tests);
    tests_expected = (uchar*) malloc(size_expected * num_tests);
    assert(tests_v1 != NULL);
    assert(tests_v2 != NULL);
    assert(tests_expected != NULL);

    // these buffers are just for reading from the file
    char* buf_v1 = (char*) malloc(num_inputs + 1);
    assert(buf_v1 != NULL);
    char* buf_v2 = (char*) malloc(num_inputs + 1);
    assert(buf_v2 != NULL);
    char* buf_expected = (char*) malloc(num_outputs + 1);
    assert(buf_expected != NULL);
    for (uint test_id = 0; test_id < num_tests; test_id++)
    {
        if (fscanf(fp, "%s %s %s\n", buf_v1, buf_v2, buf_expected) != 3)
        {
            fprintf(stderr, "Unable to parse tests line #%d!\n", test_id);
            exit(1);
        }

        // now we need to convert the values to our new special compacted binary format
        for (uint i = 0; i < num_inputs; i++)
        {
            BIT_SET_UCHAR(tests_v1 + size_v1_v2 * test_id, i, (buf_v1[i] == '0') ? 0 : 1);
            BIT_SET_UCHAR(tests_v2 + size_v1_v2 * test_id, i, (buf_v2[i] == '0') ? 0 : 1);
        }
        for (uint i = 0; i < num_outputs; i++)
        {
            BIT_SET_UCHAR(tests_expected, size_expected * test_id + i, (buf_expected[i] == '0') ? 0 : 1);
        }
    }
    free(buf_v1);
    free(buf_v2);
    free(buf_expected);
    fclose(fp);


    // Read in list of faults
    fp = fopen(filename_faults, "r");
    assert(fp != NULL);

    if (fscanf(fp, "NUM_FAULTS %d\n", &num_faults) != 1)
    {
        fprintf(stderr, "Unable to parse NUM_FAULTS line!\n");
        exit(1);
    }

    printf("Detected %d faults\n", num_faults);
    faults = (Fault*) malloc(sizeof(Fault) * num_faults);
    assert(faults != NULL);
    uint net, polarity;
    for (uint ix = 0; ix < num_faults; ix++)
    {
        if (fscanf(fp, "%d %d\n", &net, &polarity) != 2)
        {
            fprintf(stderr, "Unable to parse fault line #%d!\n", ix);
            exit(1);
        }

        faults[ix].net = net;
        faults[ix].polarity = polarity;
    }
    fclose(fp);


    printf("Finished parsing files!\n");
    printf("--------------------------------------------\n");

    // ------------- End "please move to separate function/file" section --------------

    gettimeofday(&tvDoneParsing, NULL);

    /***************************************************************************
     * The Grand Plan
     * A single-fault-multiple-pattern approach:
     * 1. First, we run a parallel fault simulation to determine the fault-free
     * circuit state for every test pair.
     * 2. Then, for each fault, we determine which test pairs activate the fault
     * 3. We can ignore test pairs that do not activate the fault.
     * 4. We do another parallel fault simulation on only the patterns that
     *    activate the fault. We also have to change each state to put the X
     *    value at the necessary net before fault simulation.
    ***************************************************************************/

    /*_  ________ _____  _   _ ______ _        __
    | |/ /  ____|  __ \| \ | |  ____| |      /_ |
    | ' /| |__  | |__) |  \| | |__  | |       | |
    |  < |  __| |  _  /| . ` |  __| | |       | |
    | . \| |____| | \ \| |\  | |____| |____   | |
    |_|\_\______|_|  \_\_| \_|______|______|  |_|
    */

    // We need to store the circuit state (v1 and v2 values for all nets) for all tests.
    uint num_nets = num_inputs + num_gates;
    uint num_state_values = num_nets * 2; // times 2, since we have v1 and v2 state! see explanation above
    uint state_bytes = num_state_values; // weird, but it works
    uint all_states_size = state_bytes * num_tests; // number of bytes required to store a state for each test
    pretty_bytes(buffer, all_states_size);
    printf("Detected %d nets, requiring %u B per state, %s total\n", num_nets, state_bytes, buffer);

    // we allocate a circuit state for each test, and then we find the fault-free values in the circuit for each test
    uchar* fault_free_states = (uchar*) malloc(all_states_size);
    assert(fault_free_states != NULL);
    // let's set the values all to X (X=01, so 01010101, so 0x55) - This uses our new compact format for storing the state information
    memset(fault_free_states, 0x55, all_states_size);
    //print_state(fault_free_states, num_state_values);
    //print_state_raw(fault_free_states, num_state_values);


    // 1. First, we run a parallel fault simulation to determine the fault-free circuit state for every test pair
    for (uint test_id = 0; test_id < num_tests; test_id++)
    {
        // set the input values in the fault free states
        uchar* this_state = fault_free_states + test_id * state_bytes;
        for (uint input_id = 0; input_id < num_inputs; input_id++)
        {
            this_state[inputs[input_id] * 2]     = (BIT_GET_UCHAR(tests_v1 + size_v1_v2 * test_id, input_id) == 0 ? LOGIC_0 : LOGIC_1);
            this_state[inputs[input_id] * 2 + 1] = (BIT_GET_UCHAR(tests_v2 + size_v1_v2 * test_id, input_id) == 0 ? LOGIC_0 : LOGIC_1);
        }
    }

    // now that we have set the input patterns in all the fault-free states, we run the simulations in parallel to find the fault-free circuit states for all tests

    // the gpu needs copies of the states and the gates list-of-structs
    uchar* dev_fault_free_states;
    hipMalloc( (void**)&dev_fault_free_states, all_states_size);
    check_cuda_errors("(hipMalloc dev_fault_free_states)");
    assert(dev_fault_free_states != NULL);
    hipMemcpy( dev_fault_free_states, fault_free_states, all_states_size, hipMemcpyHostToDevice );
    check_cuda_errors("(hipMemcpy dev_fault_free_states to GPU)");

    Gate* dev_gates;
    hipMalloc( (void**)&dev_gates, sizeof(Gate) * num_gates);
    check_cuda_errors("(hipMalloc dev_gates)");
    assert(dev_gates != NULL);
    hipMemcpy( dev_gates, gates, sizeof(Gate) * num_gates, hipMemcpyHostToDevice );
    check_cuda_errors("(hipMemcpy dev_gates to GPU)");

    // Launch Kernel 1! We have a blocks with FAULTS_PER_BLOCK_KERNEL_1 threads, one thread for each test
    gettimeofday(&tvPreK1, NULL);
    cuda_fault_free_fault_sim<<< divide_round_up(num_tests, FAULTS_PER_BLOCK_KERNEL_1), FAULTS_PER_BLOCK_KERNEL_1 >>>(dev_gates, num_gates, dev_fault_free_states, state_bytes, num_tests, use_trax);
    check_cuda_errors("1 (fault free fault simulation)");
    gettimeofday(&tvPostK1, NULL);
    printf("finished with fault-free responses kernel #1\n");


/*_  ________ _____  _   _ ______ _        ___
 | |/ /  ____|  __ \| \ | |  ____| |      |__ \
 | ' /| |__  | |__) |  \| | |__  | |         ) |
 |  < |  __| |  _  /| . ` |  __| | |        / /
 | . \| |____| | \ \| |\  | |____| |____   / /_
 |_|\_\______|_|  \_\_| \_|______|______| |____|
*/

    // Now, at this point, we have the fault-free responses for all tests
    // From the grand plan: "2. Then, for each fault, we determine which test pairs activate the fault."
    // Each thread corresponds with a single fault, and determines which tests activate the fault
    // NEW PLAN: We need to pack this data tighter using BIT_SET_UCHAR
    uint size_fault_activations = divide_round_up(num_tests * num_gates * 2, 8);
    pretty_bytes(buffer, size_fault_activations);
    printf("We need %s to store %d potential fault activation bits\n", buffer, num_tests * num_gates * 2);
    uchar* fault_activations = (uchar*) malloc(size_fault_activations);
    assert (fault_activations != NULL);
    memset(fault_activations, 0, size_fault_activations);

    uchar* dev_fault_activations;
    hipMalloc( (void**)&dev_fault_activations, size_fault_activations);
    check_cuda_errors("hipMalloc (dev_fault_activations)");
    assert(dev_fault_activations != NULL);
    // The memcpy below just copies in zeros. Is there a way to get around this, maybe an initializing cudaMalloc() ? TODO
    hipMemcpy( dev_fault_activations, fault_activations, size_fault_activations, hipMemcpyHostToDevice );
    check_cuda_errors("hipMemcpy (dev_fault_activations zeros to GPU)");

    // Each thread checks all tests to see which tests activate its fault.
    // dev_fault_free_states is still in the GPU, no need to copy it back and forth between kernels!
    gettimeofday(&tvPreK2, NULL);
    uint num_blocks_kernel_2 = divide_round_up(num_gates * 2, FAULTS_PER_BLOCK_KERNEL_2);
    cuda_check_fault_activations<<< num_blocks_kernel_2, FAULTS_PER_BLOCK_KERNEL_2 >>>(dev_gates, dev_fault_free_states, state_bytes, num_tests, dev_fault_activations, num_gates * 2);
    check_cuda_errors("2 (fault activations)");
    gettimeofday(&tvPostK2, NULL);

    hipMemcpy( fault_activations, dev_fault_activations, size_fault_activations, hipMemcpyDeviceToHost );
    check_cuda_errors("post-2 (hipMemcpy fault_activations to CPU)");


/*_  ________ _____  _   _ ______ _        ____
 | |/ /  ____|  __ \| \ | |  ____| |      |___ \
 | ' /| |__  | |__) |  \| | |__  | |        __) |
 |  < |  __| |  _  /| . ` |  __| | |       |__ <
 | . \| |____| | \ \| |\  | |____| |____   ___) |
 |_|\_\______|_|  \_\_| \_|______|______| |____/

     * 4. We do another parallel fault simulation on only the patterns that
     *    activate the fault. We also have to change each state to put the X
     *    value at the necessary net before fault simulation (sequential?). */
    // At this point each fault has some number of tests that activate the fault.
    // These are the only tests we need to further simulate.
    // NEW PLAN: We'll definitely have > 512 activations for larger circuits, so we have two options:
    // 1. Separate kernel invocations for each fault (THIS IS THE CHOICE I MADE GOING FORWARD)
    //    + Re-use the faulty states memory for each fault (don't need to allocate gigs and gigs up front)
    //    + Can take advantage of "later faults skip most data" speedups
    //    - Kernel overhead of having thousands of kernel invocations
    // 2. One gigantic kernel
    //    - All threads must process entire circuit (so they stay in sync) and lose the "later faults skip most data" speedup
    //    - Need to allocate num_activations * state_bytes bytes of memory (which can be huge)
    //    + Only one kernel invocation, so we avoid any/all overhead with kernel calls
    // As noted above, we decided to go with option 1, which seems to be working well for now.

    // Need an array of how many tests need to be run (for each fault)
    uint* num_fault_activations = (uint*) malloc(sizeof(uint) * num_gates * 2);
    assert(num_fault_activations != NULL);

    // This is the array of offsets into the big state table (for each fault)
    uint* fault_activations_offset = (uint*) malloc(sizeof(uint) * num_gates * 2);
    assert(fault_activations_offset != NULL);

    ulong total_activations = 0;
    ulong max_num_activations = 0;
    for (uint fault_id = 0; fault_id < num_gates * 2; fault_id++)
    {
        fault_activations_offset[fault_id] = total_activations;

        uint count = 0;
        for (uint test_id = 0; test_id < num_tests; test_id++)
            count += BIT_GET_UCHAR(fault_activations, fault_id * num_tests + test_id);
        num_fault_activations[fault_id] = count;
        //printf("Fault %8d activated by %8d tests\n", fault_id, count);

        total_activations += count;
        if (count > max_num_activations)
            max_num_activations = count;
    }
    printf("Max num activations: %ld\n", max_num_activations);
    //printf("------------------------------------------------\n");

    // let's make an array of the test_id values for each fault, in order for fault_0, then fault_1, etc
    // Note, we can't merge this pair of loops with the very similar pair of loops above, because we need to know total_activations before we can malloc here
    //      It's not a big deal because this part doesn't take much of the time
    uint* activating_test_ids = (uint*) malloc(sizeof(uint) * total_activations);
    assert(activating_test_ids != NULL);
    uint array_index = 0;
    for (uint fault_id = 0; fault_id < num_gates * 2; fault_id++)
    {
        for (uint test_id = 0; test_id < num_tests; test_id++)
        {
            if (BIT_GET_UCHAR(fault_activations, fault_id * num_tests + test_id))
            {
                activating_test_ids[array_index] = test_id;
                array_index += 1;
            }
        }
    }

    // NEW PLAN - Eventually we'll have > 512 activations per fault, so we have to move to a new kernel 3 architecture.
    // Let's have one kernel-3 invocation per fault, with FAULTS_PER_BLOCK_KERNEL_3 threads (tests) per block, and as many blocks as we need.
    // We don't have to have as much memory here to hold all the faulty states, just as many as the maximum number of activations for a given fault.

    // Now we have to allocate a boat-load of memory to store all the circuit states for all the faults for all tests.
    // Before, we only stored max_num_activations circuit states, but that meant we had to do a lot of tiny isolated memcopies which is really slow.
    // New plan is to allocate all num_tests circuit states, and have each thread use one of those states. Some states will not be touched
    // we only need to store max_num_activations_for_single_fault circuit states.
    pretty_bytes(buffer, all_states_size);
    printf("We require %s for our faulty states!\n", buffer);
    uchar* faulty_states = (uchar*) malloc(all_states_size);
    assert(faulty_states != NULL);
    // Since kernel 3 now is able to activate the fault (set LOGIC_X in circuit state)
    // No need to copy faulty_states to the CPU, activate the faults, and copy it back to the GPU.
    uchar* dev_faulty_states;
    hipMalloc( (void**)&dev_faulty_states, all_states_size);
    check_cuda_errors("pre-3 (hipMalloc faulty_states)");
    assert(dev_faulty_states != NULL);

    uint* dev_activating_test_ids;
    hipMalloc( (void**)&dev_activating_test_ids, sizeof(uint) * total_activations );
    check_cuda_errors("pre-3 (hipMalloc dev_activating_test_ids)");
    assert(dev_activating_test_ids != NULL);
    hipMemcpy(dev_activating_test_ids, activating_test_ids, sizeof(uint) * total_activations, hipMemcpyHostToDevice);
    check_cuda_errors("pre-3 (hipMemcpy dev_activating_test_ids into GPU)");

    uint dict_size = divide_round_up(num_faults * num_tests, 8);
    pretty_bytes(buffer, dict_size);
    printf("We require %s for the packed dictionary data (%d faults, %d tests)\n", buffer, num_faults, num_tests);
    uchar* dict = (uchar*) malloc(dict_size);
    assert(dict != NULL);
    memset(dict, 0, dict_size);

    uchar* dev_dict;
    hipMalloc( (void**)&dev_dict, dict_size );
    check_cuda_errors("pre-3 (hipMalloc dev_dict)");
    assert(dev_dict != NULL);
    hipMemcpy(dev_dict, dict, dict_size, hipMemcpyHostToDevice); // Again, we're copying all 0s from CPU to GPU, can't we just init or cudaMemset? TODO
    check_cuda_errors("pre-3 (hipMemcpy empty dict to GPU)");

    struct timeval tvStep;
    gettimeofday(&tvPreK3, NULL);
    gettimeofday(&tvStep, NULL);
    hipProfilerStart();
    for (uint fault_list_index = 0; fault_list_index < num_faults; fault_list_index++) {
        Fault *fault = &faults[fault_list_index];
        uint fault_id = (fault->net * 2) + fault->polarity;

        // it may be the case that there are no fault activations, in which case we just don't run the kernel
        if (num_fault_activations[fault_id] > 0) {
            // copy a fresh copy of the fault-free states into the faulty_states
            hipMemcpy(dev_faulty_states, dev_fault_free_states, all_states_size, hipMemcpyDeviceToDevice);
            check_cuda_errors("pre-3 (hipMemcpy dev_fault_free_states -> dev_faulty_states - GPU-to-GPU)");
            hipDeviceSynchronize();

            uint my_activations_offset = fault_activations_offset[fault_id]; // Where we have to start in the dev_activating_test_ids for this fault
            uint num_blocks = divide_round_up(num_fault_activations[fault_id], FAULTS_PER_BLOCK_KERNEL_3);
            cuda_faulty_fault_sim<<< num_blocks, FAULTS_PER_BLOCK_KERNEL_3 >>>(dev_gates, num_gates, dev_faulty_states, state_bytes, num_fault_activations[fault_id], my_activations_offset, dev_activating_test_ids, dev_dict, fault_list_index, num_tests, fault_id, use_trax);
            check_cuda_errors("3 (faulty fault sim)");
            hipDeviceSynchronize();
        }

        float progress = (fault_list_index + 1) / (1.0 * num_faults);
        struct timeval tvNow, tvTemp;
        gettimeofday(&tvNow, NULL);

        // step = tvNow - time_step
        timeval_subtract(&tvTemp, &tvNow, &tvStep);
        // time_so_far = tvNow - time_start
        timeval_subtract(&tvDiff, &tvNow, &tvPreK3);

        unsigned long int time_so_far_us = tvDiff.tv_usec + 1000000 * tvDiff.tv_sec;
        unsigned long int time_left_us = (long int)(((1 - progress) * time_so_far_us) / progress);
        printf("\rFault id %6d, %6d activations (%6d / %6d = %3.6f - %ld.%06ld total, %ld.%06ld step, %ld left)",
               fault_id, num_fault_activations[fault_id],
               fault_list_index + 1, num_faults, progress,
               tvDiff.tv_sec, tvDiff.tv_usec,
               tvTemp.tv_sec, tvTemp.tv_usec,
               time_left_us / 1000000);

        gettimeofday(&tvStep, NULL);
    }
    hipProfilerStop();
    gettimeofday(&tvPostK3, NULL);
    printf("\n");

    // now write the dictionary data to disk
    hipMemcpy(dict, dev_dict, dict_size, hipMemcpyDeviceToHost);
    check_cuda_errors("post-3 (copying dictionary to CPU)");
    fp = fopen(filename_dictionary, "w");
    for (uint fault_list_index = 0; fault_list_index < num_faults; fault_list_index++)
    {
        for (uint test_id = 0; test_id < num_tests; test_id++)
        {
            fprintf(fp, "%d", BIT_GET_UCHAR(dict, fault_list_index * num_tests + test_id));
        }
        fprintf(fp, "\n");
    }
    fclose(fp);
    gettimeofday(&tvEnd, NULL);

    //printf("----------------------------------\n");
    printf("Wrote dictionary to '%s', goodbye!\n", filename_dictionary);
    printf("----------------------------------\n");
    printf("Detailed timing information:\n");

    timeval_subtract(&tvDiff, &tvDoneParsing, &tvStart);
    printf("1. %ld.%06ld Parse the input files: \n", tvDiff.tv_sec, tvDiff.tv_usec);

    timeval_subtract(&tvDiff, &tvPreK1, &tvDoneParsing);
    printf("2. %ld.%06ld Get ready for kernel 1\n", tvDiff.tv_sec, tvDiff.tv_usec);

    timeval_subtract(&tvDiff, &tvPostK1, &tvPreK1);
    printf("3. %ld.%06ld Kernel 1\n", tvDiff.tv_sec, tvDiff.tv_usec);

    timeval_subtract(&tvDiff, &tvPreK2, &tvPostK1);
    printf("4. %ld.%06ld Get ready for kernel 2\n", tvDiff.tv_sec, tvDiff.tv_usec);

    timeval_subtract(&tvDiff, &tvPostK2, &tvPreK2);
    printf("5. %ld.%06ld Kernel 2\n", tvDiff.tv_sec, tvDiff.tv_usec);

    timeval_subtract(&tvDiff, &tvPreK3, &tvPostK2);
    printf("6. %ld.%06ld Get ready for kernel 3\n", tvDiff.tv_sec, tvDiff.tv_usec);

    timeval_subtract(&tvDiff, &tvPostK3, &tvPreK3);
    printf("7. %ld.%06ld Kernel 3 loop\n", tvDiff.tv_sec, tvDiff.tv_usec);

    timeval_subtract(&tvDiff, &tvEnd, &tvPostK3);
    printf("8. %ld.%06ld Writing dictionary to file\n", tvDiff.tv_sec, tvDiff.tv_usec);

    printf("Total time:\n");
    timeval_subtract(&tvDiff, &tvEnd, &tvStart);
    printf("   %ld.%06ld\n", tvDiff.tv_sec, tvDiff.tv_usec);


    // free up the allocated CUDA memory
    hipFree(dev_fault_free_states);
    hipFree(dev_gates);
    hipFree(dev_fault_activations);
    hipFree(dev_faulty_states);
    hipFree(dev_activating_test_ids);
    hipFree(dev_dict);

    // free up the allocated CPU memory
    // TODO double-check all these CPU-side free calls again!
    free(filename_input);
    free(filename_tests);
    free(filename_dictionary);
    free(filename_faults);
    free(inputs);
    free(outputs);
    free(gates);
    free(tests_v1);
    free(tests_v2);
    free(tests_expected);
    free(faults);

    free(fault_free_states);
    free(fault_activations);
    free(num_fault_activations);
    free(fault_activations_offset);
    free(activating_test_ids);
    free(faulty_states);
    free(dict);

    // Apparently some profiling data is transfered asyncronously so we have to call this function to wait for those transfers to finish
    hipDeviceReset();

    return 0;
}

